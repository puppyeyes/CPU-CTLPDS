#include "abpds.cuh"

Delta delta;

ABPDSInfo *abpds_info;

void initABPDSInfo() {
	hipMallocManaged(&abpds_info, sizeof(int) * 2);
}

void initDelta(int delta_size, Delta delta) {
	hipMallocManaged(&delta, sizeof(int) * delta_size);
}

int getTransitionPos(TransitionRule *t) {
	int pos = t->to->controlLocation * (abpds_info->stack_size) + t->to->stack1;
	return pos;
}

void addRuleToDelta(TransitionRule *t, Delta delta) {
	//头插法
	int pos=getTransitionPos(t);
	if(delta[pos].next==NULL)
	{
		delta[pos].next=t;
	}
	else{
		t->next=delta[pos].next;
		delta[pos].next=t;
	}
}
