#include "hip/hip_runtime.h"
#include "common.cuh"

Gqueue *gqueue;

void initGQueue(int queue_size) {
	CUDA_SAFE_CALL(hipMallocManaged(&(gqueue), sizeof(Gqueue)));
	gqueue->mutex = 0;
	gqueue->size = queue_size;
	CUDA_SAFE_CALL(
			hipMallocManaged(&(gqueue->queue),
					sizeof(TransQueue) * queue_size));
}

void add_one_to_queue(Trans t) {
	int pos = gqueue->head;
	if (gqueue->head < gqueue->size) {
		gqueue->queue[pos] = t;
		pos++;
	} else {
		printf("queue full\n");
	}
}

__device__ void d_add_one_to_queue(Trans t, Gqueue *gqueue) {
	int pos = atomicAdd(&(gqueue->head), 1);
	if (gqueue->head < gqueue->size) {
		gqueue->queue[pos] = t;
	} else {
		printf("queue full\n");
	}
}

__device__ void get_Gqueue_Mutex(Gqueue *gqueue) {
	while (true) {
		if (atomicCAS(&(gqueue->mutex), 0, 1) == 0) {
			break;
		}
	}
}
__device__ void free_Gqueue_Mutex(Gqueue *gqueue) {
	atomicCAS(&(gqueue->mutex), 1, 0);
}

__device__ __host__ void printTrans(Trans t){
	printf("%d %d --> %d\n",t.fromState,t.stack,t.toState);
}
