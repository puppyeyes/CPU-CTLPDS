#include "common.cuh"

Gqueue *gqueue;
void initGQueue(int queue_size){
	CUDA_SAFE_CALL(hipMallocManaged(&(gqueue), sizeof(Gqueue)));
	CUDA_SAFE_CALL(hipMallocManaged(&(gqueue->queue), sizeof(TransQueue)*queue_size));
}
