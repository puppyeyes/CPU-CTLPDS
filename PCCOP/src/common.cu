#include "hip/hip_runtime.h"
#include "common.cuh"

Gqueue *gqueue;

void initGQueue(int queue_size){
	CUDA_SAFE_CALL(hipMallocManaged(&(gqueue), sizeof(Gqueue)));
	gqueue->mutex=0;
	CUDA_SAFE_CALL(hipMallocManaged(&(gqueue->queue), sizeof(TransQueue)*queue_size));
}

__device__  void get_Gqueue_Mutex(Gqueue *gqueue){
	while(true){
		if(atomicCAS(&(gqueue->mutex),0,1)==0){
			break;
		}
	}
}
__device__  void free_Gqueue_Mutex(Gqueue *gqueue){
	atomicCAS(&(gqueue->mutex),1,0);
}
