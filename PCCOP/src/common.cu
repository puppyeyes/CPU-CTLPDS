#include "hip/hip_runtime.h"
#include "common.cuh"

#define FROMSTATEMASK 0xffff000000000000
#define STACKMASK 0x0000ffffffff0000
#define TOSTATEMASK 0x000000000000ffff
Gqueue *gqueue;


void initGQueue(int queue_size) {
	CUDA_SAFE_CALL(hipMallocManaged(&(gqueue), sizeof(Gqueue)));
	gqueue->mutex = 0;
	gqueue->size = queue_size;
	CUDA_SAFE_CALL(
			hipMallocManaged(&(gqueue->queue),
					sizeof(unsigned long long int) * queue_size));
}

void add_one_to_queue(Trans t) {
	int pos = gqueue->head;
	if (gqueue->head < gqueue->size) {
		gqueue->queue[pos]=encode_trans_to_long(t);
		gqueue->head++;

	} else {
		printf("queue full\n");
	}
}

__device__ __host__ unsigned long long int encode_trans_to_long(Trans t) {
	unsigned long long int res = 0;
	res=t.fromState;
	res=res<<32;
	res=res+t.stack;
	res=res<<16;
	res=res+(unsigned short)t.toState;
	return res;
}

__device__ __host__ Trans decode_long_to_trans(unsigned long long int t){
	Trans res;
	res.fromState=(short int)((t&FROMSTATEMASK)>>48);
	res.stack=(int)((t&STACKMASK)>>16);
	res.toState=(short int)(t&TOSTATEMASK);
	return res;
}

__device__ void d_add_one_to_queue(Trans t, Gqueue *gqueue) {
	int pos = atomicAdd(&(gqueue->head), 1);
	if (gqueue->head < gqueue->size) {
		gqueue->queue[pos] = encode_trans_to_long(t);
	} else {
		printf("one queue full\n");
	}
}

__device__ void get_Gqueue_Mutex(Gqueue *gqueue) {
	while (true) {
		if (atomicCAS(&(gqueue->mutex), 0, 1) == 0) {
			break;
		}
	}
}
__device__ void free_Gqueue_Mutex(Gqueue *gqueue) {
	atomicCAS(&(gqueue->mutex), 1, 0);
}

__device__ int encode_state_superScript(int state, short int recursion) {
	int res = (state&STATEMASK) | (recursion << STATEBIT);
	return res;
}

__host__ __device__ short int decode_state_superScript(int state) {
	int res = state >> STATEBIT;
	return res;
}

__device__ __host__ void printTrans(Trans t) {
	if(t.toState!=-1){
		int superScript = decode_state_superScript(t.toState);
		int toState = t.toState & STATEMASK;
		printf("%d %d --> %d[%d](%d)\n", t.fromState, t.stack, toState,
				superScript,t.toState);
	}else{
		printf("%d %d --> %d\n", t.fromState, t.stack, t.toState);
	}

}

__device__ __host__ void printGQueue(Gqueue *gqueue) {
	printf("打印queue\n");
	for (int i = 0; i < gqueue->head; i++) {
		Trans t=decode_long_to_trans(gqueue->queue[i]);
		printTrans(t);
	}
	printf("打印queue结束\n");
}
