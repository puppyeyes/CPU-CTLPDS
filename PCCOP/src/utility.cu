#include "utility.cuh"

map<string, int> state_mp;
map<int, string> rv_state_mp;
map<string, int> stack_mp;
map<int, string> rv_stack_mp;
static int parse_abpds(xmlDocPtr doc, xmlNodePtr cur) {
	int state_count = 0; //记录state的个数
	int stack_count = 0; //记录state的个数
	/*建立一個迭代器查找MAP*/
	map<string, int>::iterator it_find;

	assert(doc || cur);
	int finalStateSize;
	/*
	 * 初始化映射stack * 爲0
	 */
	stack_mp.insert(pair<string, int>("*", stack_count));
	rv_stack_mp.insert(pair<int, string>(stack_count, "*"));
	stack_count++;

	/*统计delta的大小*/
	int delta_size;
	if (!xmlStrcmp(cur->name, (const xmlChar *) "abpds")) {
		int state_size = atoi((char *) xmlGetProp(cur, BAD_CAST "state_size"));
		int stack_size = atoi((char *) xmlGetProp(cur, BAD_CAST "stack_size"));
		delta_size = state_size * state_size;
		initABPDSInfo();
		abpds_info->stack_size = stack_size+1;
		abpds_info->state_size = state_size;
	}
	initDelta(delta_size);
	/*
	 * 计算finalStateArray的存放位置
	 * */
	int finalStateCount = 0;
	cur = cur->xmlChildrenNode;
	string tmp;
	while (cur != NULL) {
		//获取final state
		if (!xmlStrcmp(cur->name, (const xmlChar *) "finalState")) {
			//printf("parse finalState\n");

			finalStateSize = atoi((char *) xmlGetProp(cur, BAD_CAST "size"));
			//printf("final state size: %d\n", finalStateSize);
			CUDA_SAFE_CALL(
					hipMallocManaged(&finalStateArray,
							sizeof(int) * finalStateSize));
			abpds_info->finalStateSize = finalStateSize;
			xmlNodePtr stateCur = cur->xmlChildrenNode;
			while (stateCur != NULL) {
				if (!xmlStrcmp(stateCur->name, (const xmlChar *) "state")) {
					tmp = (char*) xmlNodeGetContent(stateCur);
					if (!state_mp.count(tmp)) {
						state_mp.insert(pair<string, int>(tmp, state_count));
						rv_state_mp.insert(pair<int, string>(state_count, tmp));
						finalStateArray[finalStateCount] = state_count;
						finalStateCount++;
						state_count++;
					}
				}
				stateCur = stateCur->next;
			}
		}
		//获取transition rule
		if ((!xmlStrcmp(cur->name, (const xmlChar *) "delta"))) {
			printf("parse delta\n");
			delta_size = atoi((char *) xmlGetProp(cur, BAD_CAST "size"));
			xmlNodePtr ruleCur = cur->xmlChildrenNode;
			while (ruleCur != NULL) {
				if ((!xmlStrcmp(ruleCur->name, (const xmlChar *) "rule"))) {
					int to_config_size = atoi(
							(char *) xmlGetProp(ruleCur, BAD_CAST "toSize"));

					TransitionRule *r;
					CUDA_SAFE_CALL(
							hipMallocManaged(&r, sizeof(TransitionRule)));
					xmlNodePtr configCur = ruleCur->xmlChildrenNode;
					while (configCur != NULL) {
						if ((!xmlStrcmp(configCur->name, BAD_CAST "from"))) {
							string from_control_location = (char *) xmlGetProp(
									configCur,
									BAD_CAST "controlLocation");
							//cout << from_control_location;
							if (!state_mp.count(from_control_location)) {
								state_mp.insert(
										pair<string, int>(from_control_location,
												state_count));
								rv_state_mp.insert(
										pair<int, string>(state_count,
												from_control_location));
								r->from.controlLocation = state_count;
								state_count++;
							} else {
								it_find = state_mp.find(from_control_location);
								if (it_find != state_mp.end()) {
									r->from.controlLocation = it_find->second;
								} else {
									printf("xml parse error:inter error 1\n");
								}
							}
							string from_stack = (char *) xmlGetProp(configCur,
							BAD_CAST "stack");
							//cout << "," << from_stack << "-->";
							if (from_stack.compare("*") == 0) {
								printf("xml parse error:syntax error\n");
							}
							if (!stack_mp.count(from_stack)) {
								stack_mp.insert(
										pair<string, int>(from_stack,
												stack_count));
								rv_stack_mp.insert(
										pair<int, string>(stack_count,
												from_stack));
								r->from.stack = stack_count;
								stack_count++;
							} else {
								it_find = stack_mp.find(from_stack);
								if (it_find != stack_mp.end()) {
									r->from.stack = it_find->second;
								}
							}
						}

						if (to_config_size == 1) {
							r->to_config_size = to_config_size;
							if ((!xmlStrcmp(configCur->name,
									(const xmlChar *) "to"))) {
								string to_control_location =
										(char *) xmlGetProp(configCur,
										BAD_CAST "controlLocation");
								//cout << to_control_location;
								//r.to = (ToConfig *) malloc(sizeof(ToConfig));
								CUDA_SAFE_CALL(
										hipMallocManaged(&(r->to),
												sizeof(ToConfig)));
								if (!state_mp.count(to_control_location)) {
									state_mp.insert(
											pair<string, int>(
													to_control_location,
													state_count));
									rv_state_mp.insert(
											pair<int, string>(state_count,
													to_control_location));
									r->to[0].controlLocation = state_count;
									state_count++;
								} else {
									it_find = state_mp.find(
											to_control_location);
									if (it_find != state_mp.end()) {
										r->to[0].controlLocation =
												it_find->second;
									} else {
										printf(
												"xml parse error:inter error 3\n");
									}
								}
								string to_control_stack_1 = (char *) xmlGetProp(
										configCur,
										BAD_CAST "stack1");
								//cout << "," << to_control_stack_1;
								if (!stack_mp.count(to_control_stack_1)) {
									stack_mp.insert(
											pair<string, int>(
													to_control_stack_1,
													stack_count));
									rv_stack_mp.insert(
											pair<int, string>(stack_count,
													to_control_stack_1));
									r->to[0].stack1 = stack_count;
									stack_count++;
								} else {
									it_find = stack_mp.find(to_control_stack_1);
									if (it_find != stack_mp.end()) {
										r->to[0].stack1 = it_find->second;
									}
								}
								if ((char *) xmlGetProp(configCur,
								BAD_CAST "stack2") != NULL) {
									string to_control_stack_2 =
											(char *) xmlGetProp(configCur,
											BAD_CAST "stack2");
									//cout << " " << to_control_stack_2 << endl;
									if (!stack_mp.count(to_control_stack_2)) {
										stack_mp.insert(
												pair<string, int>(
														to_control_stack_2,
														stack_count));
										rv_stack_mp.insert(
												pair<int, string>(stack_count,
														to_control_stack_2));
										r->to[0].stack2 = stack_count;
										stack_count++;
									} else {
										it_find = stack_mp.find(
												to_control_stack_2);
										if (it_find != stack_mp.end()) {
											r->to[0].stack2 = it_find->second;
										}
									}
								}
							}
						} else {
							r->to_config_size = to_config_size;
							if ((!xmlStrcmp(configCur->name,
									(const xmlChar *) "to"))) {
								CUDA_SAFE_CALL(
										hipMallocManaged(&(r->to),
												sizeof(ToConfig)
														* to_config_size));
								//int toSize = sizeof(ToConfig);
								//cout<<toSize<<endl;
								int i = -1;
								while (configCur != NULL) {
									if ((!xmlStrcmp(configCur->name,
											(const xmlChar *) "to"))) {
										i++;
										string to_control_location =
												(char *) xmlGetProp(configCur,
												BAD_CAST "controlLocation");
										//cout << to_control_location;
										if (!state_mp.count(
												to_control_location)) {
											state_mp.insert(
													pair<string, int>(
															to_control_location,
															state_count));
											rv_state_mp.insert(
													pair<int, string>(
															state_count,
															to_control_location));
											r->to[i].controlLocation =
													state_count;
											state_count++;
										} else {
											it_find = state_mp.find(
													to_control_location);
											if (it_find != state_mp.end()) {
												r->to[i].controlLocation =
														it_find->second;
											}
										}
										string to_control_stack_1 =
												(char *) xmlGetProp(configCur,
												BAD_CAST "stack1");
										if (!stack_mp.count(
												to_control_stack_1)) {
											stack_mp.insert(
													pair<string, int>(
															to_control_stack_1,
															stack_count));
											rv_stack_mp.insert(
													pair<int, string>(
															stack_count,
															to_control_stack_1));
											r->to[i].stack1 = stack_count;
											stack_count++;
										} else {
											it_find = stack_mp.find(
													to_control_stack_1);
											if (it_find != stack_mp.end()) {
												r->to[i].stack1 =
														it_find->second;
											}
										}
									}
									configCur = configCur->next;
								}
								//cout << endl;
								continue;
							}
						}
						configCur = configCur->next;
					}
					addRuleToDelta(r);
				}
				ruleCur = ruleCur->next;
			}

		}
		cur = cur->next;
	}
	return 0;
}

int parse_abpds_xml(const char *file_name) {

	assert(file_name);

	xmlDocPtr doc;   //xml整个文档的树形结构
	xmlNodePtr cur;   //xml节点

//获取树形结构
	doc = xmlParseFile(file_name);
	if (doc == NULL) {
		fprintf(stderr, "Failed to parse xml file:%s\n", file_name);
		goto FAILED;
	}

//获取根节点
	cur = xmlDocGetRootElement(doc);
	if (cur == NULL) {
		fprintf(stderr, "Root is empty.\n");
		goto FAILED;
	}

	if ((xmlStrcmp(cur->name, (const xmlChar *) "abpds"))) {
		fprintf(stderr, "The root is not abpds.\n");
		goto FAILED;
	}

//遍历处理根节点的每一个子节点
//cur = cur->xmlChildrenNode;
	if ((!xmlStrcmp(cur->name, (const xmlChar *) "abpds"))) {
//        id = xmlGetProp(cur, "id");
//        printf("id:%s\t",id);
		parse_abpds(doc, cur);
	}
	xmlFreeDoc(doc);
	return 0;
	FAILED: if (doc) {
		xmlFreeDoc(doc);
	}
	return -1;
}

void printTransitionRule(TransitionRule *r) {
	map<int, string>::iterator it_find;
	string from_stack;
	string from_state;
	string to_stack1;
	string to_stack2;
	string to_state;
	it_find = rv_state_mp.find(r->from.controlLocation);
	if (it_find != rv_state_mp.end()) {
		from_state = it_find->second;
	} else {
		printf("xml parse error:inter error 7\n");
	}

	it_find = rv_stack_mp.find(r->from.stack);
	if (it_find != rv_stack_mp.end()) {
		from_stack = it_find->second;
	}
	cout << from_state << "," << from_stack << "-->";
	for (int i = 0; i < r->to_config_size; i++) {
		it_find = rv_state_mp.find(r->to[i].controlLocation);
		if (it_find != rv_state_mp.end()) {
			to_state = it_find->second;
		}

		it_find = rv_stack_mp.find(r->to[i].stack1);
		if (it_find != rv_stack_mp.end()) {
			to_stack1 = it_find->second;
		}
		cout << to_state << "," << to_stack1 << " ";
		if (r->to_config_size == 1) {
			it_find = rv_stack_mp.find(r->to[i].stack2);
			if (it_find != rv_stack_mp.end()) {
				to_stack2 = it_find->second;
			}
			if (to_stack2.compare("*") != 0) {
				cout << to_stack2;
			}
		} else {
			cout << "&";
		}
	}
	cout << endl;
}

void print_parse_result() {
	for (int i = 0; i < ((abpds_info->stack_size) * (abpds_info->state_size));
			i++) {
		TransitionRule *r = delta[i].next;
		while (r != NULL) {
			printTransitionRule(r);
			r = r->next;
		}
	}
}

void printStateMap() {
	map<int, string>::iterator it_find;
	string state;
	cout<<"state map"<<endl;
	for (int i = 0; i < abpds_info->state_size; i++) {
		it_find = rv_state_mp.find(i);
		if (it_find != rv_state_mp.end()) {
			state = it_find->second;
		}
		cout<<state<<" "<<i<<endl;
	}
	cout<<"state map end"<<endl;
}

void printStackMap() {
	map<int, string>::iterator it_find;
	string stack;
	cout<<"stack map"<<endl;
	for (int i = 0; i < abpds_info->stack_size; i++) {
		it_find = rv_stack_mp.find(i);
		if (it_find != rv_stack_mp.end()) {
			stack = it_find->second;
		}
		cout<<stack<<" "<<i<<endl;
	}
	cout<<"stack map end"<<endl;
}
