#include "utility.cuh"
#include "abpds.cuh"
map<string, int> state_mp;
map<string, int> stack_mp;
int *finalStateArray;
static int parse_abpds(xmlDocPtr doc, xmlNodePtr cur) {
	int state_count = 0; //记录state的个数
	int stack_count = 0; //记录state的个数
	/*建立一個迭代器查找MAP*/
	map<string, int>::iterator it_find;

	assert(doc || cur);
	int finalStateSize;
	/*
	 * 初始化映射stack * 爲0
	 */
	stack_mp.insert(pair<string, int>("*", stack_count));
	stack_count++;

	/*统计delta的大小*/
	int delta_size;
	if (!xmlStrcmp(cur->name, (const xmlChar *) "abpds")) {
		int state_size = atoi((char *) xmlGetProp(cur, BAD_CAST "state_size"));
		int stack_size = atoi((char *) xmlGetProp(cur, BAD_CAST "stack_size"));
		delta_size = state_size * state_size;
	}
	initDelta(delta_size, delta);
	/*
	 * 计算finalStateArray的存放位置
	 * */
	int finalStateCount = 0;
	/*临时在cpu中存放rule方便处理*/
	TransitionRule *transitionRuleList;
	//printf("name %s\n", cur->name);
	cur = cur->xmlChildrenNode;
	string tmp;
	while (cur != NULL) {
		//获取final state
		if (!xmlStrcmp(cur->name, (const xmlChar *) "finalState")) {
			printf("parse finalState\n");

			finalStateSize = atoi((char *) xmlGetProp(cur, BAD_CAST "size"));
			printf("final state size: %d\n", finalStateSize);
			hipMallocManaged(&finalStateArray, sizeof(int) * finalStateSize);
			xmlNodePtr stateCur = cur->xmlChildrenNode;
			while (stateCur != NULL) {
				if (!xmlStrcmp(stateCur->name, (const xmlChar *) "state")) {
					tmp = (char*) xmlNodeGetContent(stateCur);
					if (!state_mp.count(tmp)) {
						state_mp.insert(pair<string, int>(tmp, state_count));
						finalStateArray[finalStateCount] = state_count;
						finalStateCount++;
						state_count++;
						cout << tmp << endl;
					}
				}
				stateCur = stateCur->next;
			}
		}
		//获取transition rule
		if ((!xmlStrcmp(cur->name, (const xmlChar *) "delta"))) {
			printf("parse delta\n");
			delta_size = atoi((char *) xmlGetProp(cur, BAD_CAST "size"));
			xmlNodePtr ruleCur = cur->xmlChildrenNode;
			while (ruleCur != NULL) {
				if ((!xmlStrcmp(ruleCur->name, (const xmlChar *) "rule"))) {
					int to_config_size = atoi(
							(char *) xmlGetProp(ruleCur, BAD_CAST "toSize"));

					TransitionRule r;
					xmlNodePtr configCur = ruleCur->xmlChildrenNode;
					while (configCur != NULL) {
						if ((!xmlStrcmp(configCur->name, BAD_CAST "from"))) {
							string from_control_location = (char *) xmlGetProp(
									configCur,
									BAD_CAST "controlLocation");
							cout << from_control_location;
							if (!state_mp.count(from_control_location)) {
								state_mp.insert(
										pair<string, int>(from_control_location,
												state_count));
								r.from.controlLocation = state_count;
								state_count++;
							} else {
								it_find = state_mp.find(from_control_location);
								if (it_find != state_mp.end()) {
									r.from.controlLocation = it_find->second;
								} else {
									printf("xml parse error:inter error 1\n");
								}
							}
							string from_stack = (char *) xmlGetProp(configCur,
							BAD_CAST "stack");
							cout << "," << from_stack << "-->";
							if (from_stack.compare("*") == 0) {
								printf("xml parse error:syntax error\n");
							}
							if (!stack_mp.count(from_stack)) {
								stack_mp.insert(
										pair<string, int>(from_stack,
												stack_count));
								r.from.stack = stack_count;
								stack_count++;
							} else {
								it_find = stack_mp.find(from_stack);
								if (it_find != stack_mp.end()) {
									r.from.stack = it_find->second;
								} else {
									printf("xml parse error:inter error 2\n");
								}
							}
						}

						if (to_config_size == 1) {
							r.tag = false;
							if ((!xmlStrcmp(configCur->name,
									(const xmlChar *) "to"))) {
								string to_control_location =
										(char *) xmlGetProp(configCur,
										BAD_CAST "controlLocation");
								cout << to_control_location;
								r.to = (ToConfig *) malloc(sizeof(ToConfig));
								if (!state_mp.count(to_control_location)) {
									state_mp.insert(
											pair<string, int>(
													to_control_location,
													state_count));
									r.to[0].controlLocation = state_count;
									state_count++;
								} else {
									it_find = state_mp.find(
											to_control_location);
									if (it_find != state_mp.end()) {
										r.to[0].controlLocation =
												it_find->second;
									} else {
										printf(
												"xml parse error:inter error 3\n");
									}
								}
								string to_control_stack_1 = (char *) xmlGetProp(
										configCur,
										BAD_CAST "stack1");
								cout << "," << to_control_stack_1;
								if (!stack_mp.count(to_control_stack_1)) {
									stack_mp.insert(
											pair<string, int>(
													to_control_stack_1,
													stack_count));
									r.to[0].stack1 = stack_count;
									stack_count++;
								} else {
									it_find = stack_mp.find(to_control_stack_1);
									if (it_find != stack_mp.end()) {
										r.to[0].stack1 = it_find->second;
									} else {
										printf(
												"xml parse error:inter error 4\n");
									}
								}
								if ((char *) xmlGetProp(configCur,
								BAD_CAST "stack2") != NULL) {
									string to_control_stack_2 =
											(char *) xmlGetProp(configCur,
											BAD_CAST "stack2");
									cout << " " << to_control_stack_2 << endl;
									if (!stack_mp.count(to_control_stack_2)) {
										stack_mp.insert(
												pair<string, int>(
														to_control_stack_2,
														stack_count));
										r.to[0].stack2 = stack_count;
										stack_count++;
									} else {
										it_find = stack_mp.find(
												to_control_stack_2);
										if (it_find != stack_mp.end()) {
											r.to[0].stack2 = it_find->second;
										} else {
											printf(
													"xml parse error:inter error 5\n");
										}
									}
								} else {
									cout << endl;
								}

							}
						} else {
							r.tag = true;
							if ((!xmlStrcmp(configCur->name,
									(const xmlChar *) "to"))) {
								r.to = (ToConfig *) malloc(
										sizeof(ToConfig) * to_config_size);
								int i = 0;
								while (configCur != NULL) {
									if ((!xmlStrcmp(configCur->name,
											(const xmlChar *) "to"))) {
										string to_control_location =
												(char *) xmlGetProp(configCur,
												BAD_CAST "controlLocation");
										cout << to_control_location;
										if (!state_mp.count(
												to_control_location)) {
											state_mp.insert(
													pair<string, int>(
															to_control_location,
															state_count));
											r.to[i].controlLocation =
													state_count;
											state_count++;
										} else {
											it_find = state_mp.find(
													to_control_location);
											if (it_find != state_mp.end()) {
												r.to[i].controlLocation =
														it_find->second;
											} else {
												printf(
														"xml parse error:inter error 6\n");
											}
										}
										string to_control_stack_1 =
												(char *) xmlGetProp(configCur,
												BAD_CAST "stack1");
										cout << "," << to_control_stack_1<<"&";
										if (!stack_mp.count(
												to_control_stack_1)) {
											stack_mp.insert(
													pair<string, int>(
															to_control_stack_1,
															stack_count));
											r.to[0].stack1 = stack_count;
											stack_count++;
										} else {
											it_find = stack_mp.find(
													to_control_stack_1);
											if (it_find != stack_mp.end()) {
												r.to[0].stack1 =
														it_find->second;
											} else {
												printf(
														"xml parse error:inter error 7\n");
											}
										}
									}
									configCur = configCur->next;
									i++;
								}
								cout<<endl;
								continue;
							}
						}
						configCur = configCur->next;
					}

				}
				ruleCur = ruleCur->next;
			}

		}
		cur = cur->next;
	}
	return 0;
}

int parse_abpds_xml(const char *file_name) {

	assert(file_name);

	xmlDocPtr doc;   //xml整个文档的树形结构
	xmlNodePtr cur;   //xml节点

//获取树形结构
	doc = xmlParseFile(file_name);
	if (doc == NULL) {
		fprintf(stderr, "Failed to parse xml file:%s\n", file_name);
		goto FAILED;
	}

//获取根节点
	cur = xmlDocGetRootElement(doc);
	if (cur == NULL) {
		fprintf(stderr, "Root is empty.\n");
		goto FAILED;
	}

	if ((xmlStrcmp(cur->name, (const xmlChar *) "abpds"))) {
		fprintf(stderr, "The root is not abpds.\n");
		goto FAILED;
	}

//遍历处理根节点的每一个子节点
//cur = cur->xmlChildrenNode;
	if ((!xmlStrcmp(cur->name, (const xmlChar *) "abpds"))) {
//        id = xmlGetProp(cur, "id");
//        printf("id:%s\t",id);
		parse_abpds(doc, cur);
	}
	xmlFreeDoc(doc);
	return 0;
	FAILED: if (doc) {
		xmlFreeDoc(doc);
	}
	return -1;
}
