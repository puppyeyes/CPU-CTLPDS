#include "hip/hip_runtime.h"
#include "abpdspre.cuh"
#include <hip/hip_cooperative_groups.h>

using namespace std;
using namespace cooperative_groups;
__global__ void compute_pre_on_pds(int*finish, Delta delta, AMA *ama1,
		AMA *ama2, int *finalStateArray, Gqueue *gqueue, ABPDSInfo abpds_info) {
	grid_group grid = this_grid();
	int thread_num = threadIdx.x;

	__shared__ Trans queue[32][64];
	__shared__ int count[32];
	__shared__ int start[32];
	__shared__ bool isDone;

	while (true) {
		if (thread_num == 0) {
			isDone = true;
		}






		__syncthreads();
		if ((gqueue->head - gqueue->tail == 0) && isDone) {
			atomicExch(finish, 1);
		}
		if(!isDone){
			start[0]=0;
			for(int i=1;i<32;i++)
			{
				count[i]=count[i-1]+start[0];
			}
		}
		__syncthreads();

		int thread_start_offset=start[thread_num];
		int thread_count=count[thread_num];

		for(int i=0;i<thread_count;i++){
			gqueue->queue[gqueue->head+thread_start_offset+i]=queue[thread_num][i];
		}

		if (*finish == 1) {
			grid.sync();
			if ((gqueue->head - gqueue->tail == 0)) {
				break;
			} else {
				atomicExch(finish, 0);
			}
		}

	}

}

