#include "hip/hip_runtime.h"
#include "abpdspre.cuh"
#include <hip/hip_cooperative_groups.h>

using namespace std;
using namespace cooperative_groups;

//extern __shared__ Trans queue[32][64];

__device__ inline void compute(Delta delta, AMA *latest_ama, Pool *pool,
		Trans t, ABPDSInfo *abpds_info, int thread_num, Trans queue[32][64]) {
	TransitionRule *tmp_rule = delta[t.fromState * abpds_info->stack_size
			+ t.stack].next;
	int queue_count = 0;
	while (tmp_rule != NULL) {

		if (tmp_rule->to_config_size == 1) {
			if (tmp_rule->to->stack1 == 0) {
				//<p,r>--><p,*>跳过不做
				continue;
			} else if (tmp_rule->to->stack2 == 0) {
				Trans new_t = { tmp_rule->from.controlLocation,
						tmp_rule->from.stack, t.toState };
				//<p,r>--><p,r>
				if (!isTransInAMA(new_t, latest_ama,abpds_info)) {
					d_insertTransToAMA(new_t, latest_ama, pool,abpds_info);
					printTrans(new_t);
					queue[thread_num][queue_count] = new_t;
					queue_count++;
				}
			} else {
				//<p,r>--><p,r,r>
				int tmp_state = t.toState;
				AMANode *tmp_node =
						latest_ama->list[tmp_state * abpds_info->state_size
								+ tmp_rule->to->stack2].head.next;
				while (tmp_node != NULL) {
					Trans new_t = { tmp_rule->from.controlLocation,
							tmp_rule->from.stack, tmp_node->state };
					if (!isTransInAMA(new_t, latest_ama,abpds_info)) {
						d_insertTransToAMA(new_t, latest_ama, pool,abpds_info);
						printTrans(new_t);
						queue[thread_num][queue_count] = new_t;
						queue_count++;
					}
					tmp_node = tmp_node->next;
				}
			}
		} else {
			//<p,r>--><p,r>&<p,r>...
			int isSatify = true;
			for (int i = 0; i < tmp_rule->to_config_size; i++) {
				if (!latest_ama->list[tmp_rule->to[i].controlLocation
						* abpds_info->state_size + tmp_rule->to[i].stack1].head.next
						!= NULL) {
					isSatify = false;
					break;
				}
			}
			if (isSatify) {
				for (int i = 0; i < tmp_rule->to_config_size; i++) {
					AMANode *tmp_node =
							latest_ama->list[tmp_rule->to[i].controlLocation
									* abpds_info->state_size
									+ tmp_rule->to[i].stack1].head.next;
					while (tmp_node != NULL) {
						int new_to_state = tmp_node->state;
						Trans new_t = { tmp_rule->from.controlLocation,
								tmp_rule->from.stack, new_to_state };
						if (!isTransInAMA(new_t, latest_ama,abpds_info)) {
							d_insertTransToAMA(new_t, latest_ama, pool,abpds_info);
							printTrans(new_t);
							queue[thread_num][queue_count] = new_t;
							queue_count++;
						}
						tmp_node = tmp_node->next;
					}
				}
			}
		}

		tmp_rule = tmp_rule->next;
	}
}
;

__global__ void compute_pre_on_pds(int*finish, Delta delta, AMA *latest_ama,
		int *finalStateArray, Gqueue *gqueue, ABPDSInfo *abpds_info,
		Pool *pool) {
	grid_group grid = this_grid();
	int thread_num = threadIdx.x;

	__shared__ Trans queue[32][64];
	__shared__ int count[32];
	__shared__ int offset[32];
	__shared__ bool isDone;

	while (true) {
		if (thread_num == 0) {
			isDone = true;
			get_Gqueue_Mutex(gqueue);
		}
		int active_thread_num = gqueue->head - gqueue->tail;
		//初始化 offset[] count[]
		count[thread_num] = 0;
		offset[thread_num] = 0;
		if (thread_num < active_thread_num) {
			//取出一条边
			Trans t = gqueue->queue[gqueue->head-thread_num-1];
//			printf("取出");
//			printTrans(t);
			if (thread_num == 0) {
				atomicSub(&(gqueue->head), active_thread_num);
				free_Gqueue_Mutex(gqueue);
			}
			//计算
			compute(delta, latest_ama, pool, t, abpds_info, thread_num, queue);
		}

		//计算完成之后进行同步
		__syncthreads();
		if ((gqueue->head - gqueue->tail == 0) && isDone) {
			atomicExch(finish, 1);
		}
		if (!isDone) {
			offset[0] = 0;
			for (int i = 1; i < 32; i++) {
				count[i] = count[i - 1] + offset[0];
			}
		}
		__syncthreads();
		if (!isDone) {
			if(thread_num==0){
				get_Gqueue_Mutex(gqueue);
			}
			int thread_offset = offset[thread_num];
			int thread_count = count[thread_num];

			//将计算结果加入queue
			for (int i = 0; i < thread_count; i++) {
				gqueue->queue[gqueue->head + thread_offset + i] =
						queue[thread_num][i];
			}
			if(thread_num==0){
				atomicAdd(&(gqueue->head),count[31]+offset[31]);
				free_Gqueue_Mutex(gqueue);
						}
		}
		if (*finish == 1) {
			grid.sync();
			if ((gqueue->head - gqueue->tail == 0)) {
				break;
			} else {
				atomicExch(finish, 0);
			}
		}

	}
}

__global__ void compute_epsilon(Delta delta, AMA *ama, Pool *pool,
		ABPDSInfo *abpds_info, Gqueue *gqueue) {
	int thread_num = threadIdx.x + blockIdx.x * blockDim.x;
	if (thread_num < abpds_info->state_size) {
		TransitionRule *r_h = delta[thread_num * abpds_info->stack_size].next;
		while (r_h != NULL) {
			if (r_h->to_config_size == 1) {
				Trans new_t = { r_h->from.controlLocation, r_h->from.stack,
						r_h->to[0].controlLocation };
				d_add_one_to_queue(new_t, gqueue);
				if (!isTransInAMA(new_t, ama,abpds_info)) {
					d_insertTransToAMA(new_t, ama, pool,abpds_info);
					//printTrans(new_t);
				}
			}
			r_h = r_h->next;
		}
	}
}

