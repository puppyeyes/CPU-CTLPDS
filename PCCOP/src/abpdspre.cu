#include "hip/hip_runtime.h"
#include "abpdspre.cuh"
#include <hip/hip_cooperative_groups.h>

using namespace std;
using namespace cooperative_groups;

//extern __shared__ Trans queue[32][64];

__device__ inline int compute(Delta delta, AMA *pre_ama, Pool *pool_1,
		AMA *latest_ama, Pool *pool_2, int recursion, Trans t,
		ABPDSInfo *abpds_info, int thread_num, Trans queue[32][64],
		int count[32], TMPAMA tmp_ama) {
	int isDone = 1;
	TransitionRule *tmp_rule = delta[t.fromState * abpds_info->stack_size
			+ t.stack].next;
//	printf("--------\n");
//	printRule(tmp_rule);
//	printf("--------\n");
	int queue_count = 0;
	while (tmp_rule != NULL) {
//		printRule(tmp_rule);
		if (tmp_rule->to_config_size == 1) {
			if (tmp_rule->to->stack1 == 0) {
				//<p,r>--><p,*>跳过不做
				tmp_rule = tmp_rule->next;
				continue;
			} else if (tmp_rule->to->stack2 == 0) {
				Trans new_t = { tmp_rule->from.controlLocation,
						tmp_rule->from.stack, t.toState };
				//<p,r>--><p,r>
				//printTrans(new_t);
				if (d_insertTransToAMA(new_t, latest_ama, pool_2, abpds_info)) {
					//printTrans(new_t);
					if(t.fromState==3&&t.stack==2&&t.toState==1024){
						printf("test3\n");
					}
					queue[thread_num][queue_count] = new_t;
					queue_count++;
					isDone = 0;
				}
			} else {
				//<p,r>--><p,r,r>
				int tmp_state = t.toState & STATEMASK;
				if (t.toState == -1) {
					Trans new_t = { tmp_rule->from.controlLocation,
							tmp_rule->from.stack, -1 };
					if (d_insertTransToAMA(new_t, latest_ama, pool_2,
							abpds_info)) {
						//printTrans(new_t);
						queue[thread_num][queue_count] = new_t;
						queue_count++;
						isDone = 0;
					}
				} else {
					int to_state_superScript = decode_state_superScript(
							t.toState);
					if (to_state_superScript < recursion) {
						AMANode *tmp_node =
								pre_ama->list[tmp_state * abpds_info->stack_size
										+ tmp_rule->to->stack2].head.next;
						while (tmp_node != NULL) {
							Trans new_t = { tmp_rule->from.controlLocation,
									tmp_rule->from.stack, tmp_node->state };
							if (d_insertTransToAMA(new_t, latest_ama, pool_2,
									abpds_info)) {
								//printTrans(new_t);
								queue[thread_num][queue_count] = new_t;
								queue_count++;
								isDone = 0;
							}
							tmp_node = tmp_node->next;
						}
					} else {
						AMANode *tmp_node =
								latest_ama->list[tmp_state
										* abpds_info->stack_size
										+ tmp_rule->to->stack2].head.next;
						while (tmp_node != NULL) {
							Trans new_t = { tmp_rule->from.controlLocation,
									tmp_rule->from.stack, tmp_node->state };
							if (d_insertTransToAMA(new_t, latest_ama, pool_2,
									abpds_info)) {
								//printTrans(new_t);
								queue[thread_num][queue_count] = new_t;
								queue_count++;
								isDone = 0;
							}
							tmp_node = tmp_node->next;
						}
					}

				}

			}
		} else {
			//<p,r>--><p,r>&<p,r>...
			//printTrans(t);
			bool isSatify = true;
			for (int i = 0; i < tmp_rule->to_config_size; i++) {
				int pos = tmp_rule->to[i].controlLocation
						* abpds_info->stack_size + tmp_rule->to[i].stack1;
				if ((latest_ama->list[pos].head.next == NULL)
						&& (tmp_ama[pos].next == NULL)) {
					isSatify = false;
					break;
				}
			}
			if (isSatify) {
				for (int i = 0; i < tmp_rule->to_config_size; i++) {
					int pos = tmp_rule->to[i].controlLocation
							* abpds_info->stack_size + tmp_rule->to[i].stack1;
					AMANode *tmp_node = latest_ama->list[pos].head.next;
					while (tmp_node != NULL) {
						int new_to_state = tmp_node->state;
						Trans new_t = { tmp_rule->from.controlLocation,
								tmp_rule->from.stack, new_to_state };
						if (d_insertTransToAMA(new_t, latest_ama, pool_2,
								abpds_info)) {
							//printTrans(new_t);
							queue[thread_num][queue_count] = new_t;
							queue_count++;
							isDone = 0;
						}
						tmp_node = tmp_node->next;
					}
					tmp_node = tmp_ama[pos].next;
					while (tmp_node != NULL) {
						int new_to_state = tmp_node->state;
						Trans new_t = { tmp_rule->from.controlLocation,
								tmp_rule->from.stack, new_to_state };
						if (d_insertTransToAMA(new_t, latest_ama, pool_2,
								abpds_info)) {
							//printTrans(new_t);
							queue[thread_num][queue_count] = new_t;
							queue_count++;
							isDone = 0;
						}
						tmp_node = tmp_node->next;
					}
				}
			}
		}
		tmp_rule = tmp_rule->next;
	}
	count[thread_num] = queue_count;
	return isDone;
}

__global__ void compute_pre_on_pds(int*finish, Delta delta, AMA *pre_ama,
		Pool *pool_1, AMA *latest_ama, Pool *pool_2, short int *recursion,
		Gqueue *gqueue, ABPDSInfo *abpds_info, TMPAMA tmp_ama) {
	grid_group grid = this_grid();
	int thread_num = threadIdx.x;

	__shared__ Trans queue[32][64];
	__shared__ int count[32];
	__shared__ int offset[32];
	__shared__ int isDone;

	while (true) {
		isDone = true;
		int active_thread_num = gqueue->head - gqueue->tail;
		//初始化 offset[] count[]
		count[thread_num] = 0;
		offset[thread_num] = 0;
		if (thread_num == 0&&active_thread_num>0) {
			get_Gqueue_Mutex(gqueue);
			//printf("取\n");
		}
		active_thread_num = gqueue->head - gqueue->tail;
		if(active_thread_num==0)
		{
			free_Gqueue_Mutex(gqueue);
		}
		if (thread_num < active_thread_num) {

			//取出一条边
			Trans t = decode_long_to_trans(
					gqueue->queue[gqueue->head - thread_num - 1]);
			//printTrans(t);
			__syncthreads();
			if (thread_num == 0) {
				atomicSub(&(gqueue->head), active_thread_num);
				//printf("------\n");
				free_Gqueue_Mutex(gqueue);
			}
			//计算

			atomicAnd(&isDone,
					compute(delta, pre_ama, pool_1, latest_ama, pool_2,
							*recursion, t, abpds_info, thread_num, queue, count,
							tmp_ama));
		}
		//计算完成之后进行同步
		__syncthreads();
		if ((gqueue->head - gqueue->tail == 0) && isDone) {
			atomicExch(finish, 1);
		}
		if (!isDone) {
			offset[0] = 0;
			for (int i = 1; i < 32; i++) {
				offset[i] = offset[i - 1] + count[i - 1];
			}
		}
		__syncthreads();
		if (!isDone) {
			if (thread_num == 0) {
				get_Gqueue_Mutex(gqueue);
			}
			int thread_offset = offset[thread_num];
			int thread_count = count[thread_num];

			//将计算结果加入queue
			for (int i = 0; i < thread_count; i++) {
				unsigned long long int new_t = 0;
				new_t = encode_trans_to_long(queue[thread_num][i]);
				atomicExch(&(gqueue->queue[gqueue->head + thread_offset + i]),
						new_t);
//				printf("thread:%d count:%d\n",thread_num,thread_count);
//				printTrans(gqueue->queue[gqueue->head + thread_offset + i]);
			}

			if (thread_num == 0) {
				atomicAdd(&(gqueue->head), count[31] + offset[31]);
				free_Gqueue_Mutex(gqueue);
			}
		}
		if (*finish == 1) {
			grid.sync();
			if ((gqueue->head - gqueue->tail == 0)) {
				break;
			} else {
				atomicExch(finish, 0);
			}
		}

	}
}

__global__ void compute_epsilon(Delta delta, AMA *ama, Pool *pool,
		ABPDSInfo *abpds_info, Gqueue *gqueue, short int *recursion) {
	int thread_num = threadIdx.x + blockIdx.x * blockDim.x;
	if (thread_num < abpds_info->state_size) {
		TransitionRule *r_h = delta[thread_num * abpds_info->stack_size].next;
		while (r_h != NULL) {
			if (r_h->to_config_size == 1) {
				int to_controlLocation = encode_state_superScript(
						r_h->to[0].controlLocation, *recursion);
				Trans new_t = { r_h->from.controlLocation, r_h->from.stack,
						to_controlLocation };

				if (d_insertTransToAMA(new_t, ama, pool, abpds_info)) {
					d_add_one_to_queue(new_t, gqueue);
				}
				//printTrans(new_t);
			}
			r_h = r_h->next;
		}
	}
}

