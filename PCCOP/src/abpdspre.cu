#include "hip/hip_runtime.h"
#include "abpdspre.cuh"
#include <hip/hip_cooperative_groups.h>

using namespace std;
using namespace cooperative_groups;

//extern __shared__ Trans queue[32][64];

__device__ inline bool compute(Delta delta, AMA *latest_ama, Pool *pool,
		Trans t, ABPDSInfo *abpds_info, int thread_num, Trans queue[32][64],int count[32]) {
	bool isDone=true;
	TransitionRule *tmp_rule = delta[t.fromState * abpds_info->stack_size
			+ t.stack].next;
	printf("--------\n");
	printRule(tmp_rule);
	printf("--------\n");
	int queue_count = 0;
	while (tmp_rule != NULL) {
		if (tmp_rule->to_config_size == 1) {
			if (tmp_rule->to->stack1 == 0) {
				//<p,r>--><p,*>跳过不做
				continue;
			} else if (tmp_rule->to->stack2 == 0) {
				Trans new_t = { tmp_rule->from.controlLocation,
						tmp_rule->from.stack, t.toState };
				//<p,r>--><p,r>
				if (d_insertTransToAMA(new_t, latest_ama, pool, abpds_info)) {
					//printTrans(new_t);
					queue[thread_num][queue_count].fromState = new_t.fromState;
					queue[thread_num][queue_count].stack = new_t.stack;
					queue[thread_num][queue_count].toState = new_t.toState;
					queue_count++;
					isDone=false;
				}
			} else {
				//<p,r>--><p,r,r>
				int tmp_state = t.toState;
				if (t.toState == -1) {
					Trans new_t = { tmp_rule->from.controlLocation,
							tmp_rule->from.stack, -1 };
					if (d_insertTransToAMA(new_t, latest_ama, pool,
							abpds_info)) {
						//printTrans(new_t);
						queue[thread_num][queue_count].fromState = new_t.fromState;
						queue[thread_num][queue_count].stack = new_t.stack;
						queue[thread_num][queue_count].toState = new_t.toState;
						queue_count++;
						isDone=false;
					}
				} else {
					AMANode *tmp_node =
							latest_ama->list[tmp_state * abpds_info->state_size
									+ tmp_rule->to->stack2].head.next;
					while (tmp_node != NULL) {
						Trans new_t = { tmp_rule->from.controlLocation,
								tmp_rule->from.stack, tmp_node->state };
						if (d_insertTransToAMA(new_t, latest_ama, pool,
								abpds_info)) {
							//printTrans(new_t);
							queue[thread_num][queue_count].fromState = new_t.fromState;
							queue[thread_num][queue_count].stack = new_t.stack;
							queue[thread_num][queue_count].toState = new_t.toState;
							queue_count++;
							isDone=false;
						}
						tmp_node = tmp_node->next;
					}
				}

			}
		} else {
			//<p,r>--><p,r>&<p,r>...
			int isSatify = true;
			for (int i = 0; i < tmp_rule->to_config_size; i++) {
				if (!latest_ama->list[tmp_rule->to[i].controlLocation
						* abpds_info->state_size + tmp_rule->to[i].stack1].head.next
						!= NULL) {
					isSatify = false;
					break;
				}
			}
			if (isSatify) {
				for (int i = 0; i < tmp_rule->to_config_size; i++) {
					AMANode *tmp_node =
							latest_ama->list[tmp_rule->to[i].controlLocation
									* abpds_info->state_size
									+ tmp_rule->to[i].stack1].head.next;
					while (tmp_node != NULL) {
						int new_to_state = tmp_node->state;
						Trans new_t = { tmp_rule->from.controlLocation,
								tmp_rule->from.stack, new_to_state };
						if (d_insertTransToAMA(new_t, latest_ama, pool,
								abpds_info)) {
							//printTrans(new_t);
							queue[thread_num][queue_count].fromState = new_t.fromState;
							queue[thread_num][queue_count].stack = new_t.stack;
							queue[thread_num][queue_count].toState = new_t.toState;
							queue_count++;
							isDone=false;
						}
						tmp_node = tmp_node->next;
					}
				}
			}
		}

		tmp_rule = tmp_rule->next;
	}
	count[thread_num]=queue_count;
	return isDone;
}

__global__ void compute_pre_on_pds(int*finish, Delta delta, AMA *latest_ama,
		short int *recursion, Gqueue *gqueue, ABPDSInfo *abpds_info,
		Pool *pool) {
	grid_group grid = this_grid();
	int thread_num = threadIdx.x;

	__shared__ Trans queue[32][64];
	__shared__ int count[32];
	__shared__ int offset[32];
	__shared__ bool isDone;

	while (true) {
		if (thread_num == 0) {
			isDone = true;
			get_Gqueue_Mutex(gqueue);
		}
		int active_thread_num = gqueue->head - gqueue->tail;
		//初始化 offset[] count[]
		count[thread_num] = 0;
		offset[thread_num] = 0;
		if (thread_num < active_thread_num) {
			//取出一条边
			Trans t = gqueue->queue[gqueue->head - thread_num - 1];
			printTrans(t);
			if (thread_num == 0) {
				atomicSub(&(gqueue->head), active_thread_num);
				free_Gqueue_Mutex(gqueue);
			}
			//计算
			isDone=isDone&&compute(delta, latest_ama, pool, t, abpds_info, thread_num, queue,count);
		}
		//计算完成之后进行同步
		__syncthreads();
		if ((gqueue->head - gqueue->tail == 0) && isDone) {
			atomicExch(finish, 1);
		}
		if (!isDone) {
			offset[0] = 0;
			for (int i = 1; i < 32; i++) {
				count[i] = count[i - 1] + offset[0];
			}
		}
		__syncthreads();
		if (!isDone) {
			if (thread_num == 0) {
				get_Gqueue_Mutex(gqueue);
			}
			int thread_offset = offset[thread_num];
			int thread_count = count[thread_num];

			//将计算结果加入queue
			for (int i = 0; i < thread_count; i++) {
				gqueue->queue[gqueue->head + thread_offset + i].fromState =
						queue[thread_num][i].fromState;
				gqueue->queue[gqueue->head + thread_offset + i].stack =
						queue[thread_num][i].stack;
				gqueue->queue[gqueue->head + thread_offset + i].toState =
						queue[thread_num][i].toState;
				printTrans(gqueue->queue[0]);
			}

			if (thread_num == 0) {
				atomicAdd(&(gqueue->head), count[31] + offset[31]);
				free_Gqueue_Mutex(gqueue);
			}
		}
		if (*finish == 1) {
			grid.sync();
			if ((gqueue->head - gqueue->tail == 0)) {
				break;
			} else {
				atomicExch(finish, 0);
			}
		}

	}
}

__global__ void compute_epsilon(Delta delta, AMA *ama, Pool *pool,
		ABPDSInfo *abpds_info, Gqueue *gqueue, short int *recursion) {
	int thread_num = threadIdx.x + blockIdx.x * blockDim.x;
	if (thread_num < abpds_info->state_size) {
		TransitionRule *r_h = delta[thread_num * abpds_info->stack_size].next;
		while (r_h != NULL) {
			if (r_h->to_config_size == 1) {
				encode_state_superScript(r_h->to[0].controlLocation,
						*recursion);
				Trans new_t = { r_h->from.controlLocation, r_h->from.stack,
						r_h->to[0].controlLocation };
				if (d_insertTransToAMA(new_t, ama, pool, abpds_info)) {
					d_add_one_to_queue(new_t, gqueue);
				}
				//printTrans(new_t);
			}
			r_h = r_h->next;
		}
	}
}

