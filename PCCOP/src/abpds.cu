#include "abpds.cuh"

Delta delta;

ABPDSInfo *abpds_info;

int *finalStateArray;

void initABPDSInfo() {
	hipMallocManaged(&abpds_info, sizeof(int) * 2);
}

void initDelta(int delta_size) {
	hipMallocManaged(&delta, sizeof(int) * delta_size);
}

int getTransitionPos(TransitionRule *t) {
	int pos = t->to[0].controlLocation * (abpds_info->stack_size)
			+ t->to[0].stack1;
	return pos;
}

void addRuleToDelta(TransitionRule *t) {

	//头插法
	int pos = getTransitionPos(t);
	if (delta[pos].next == NULL) {
		delta[pos].next = t;
	} else {
		t->next = delta[pos].next;
		delta[pos].next = t;
	}

	if (t->to_config_size > 1) {
		for (int i = 1; i < t->to_config_size; i++) {
			int pos = t->to[i].controlLocation * (abpds_info->stack_size)
					+ t->to[i].stack1;
			TransitionRule *t_copy;
			hipMallocManaged(&t_copy,sizeof(t));
			t_copy=t;
			t_copy->next=NULL;
			if (delta[pos].next == NULL) {
				delta[pos].next = t_copy;
			} else {
				t_copy->next = delta[pos].next;
				delta[pos].next = t_copy;
			}
		}
	}

}
__device__ void printRule(TransitionRule *rule) {
	if (rule != NULL) {
		if (rule->to_config_size == 1) {
			printf("<%d,%d>--><%d,%d %d>\n", rule->from.controlLocation,
					rule->from.stack, rule->to[0].controlLocation,
					rule->to[0].stack1, rule->to[0].stack2);
		}
	} else {
		printf("no rule\n");
	}
}

