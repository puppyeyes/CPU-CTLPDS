#include "abpds.cuh"

Delta delta;

ABPDSInfo *abpds_info;

int *finalStateArray;
void initABPDSInfo() {
	hipMallocManaged(&abpds_info, sizeof(int) * 2);
}

void initDelta(int delta_size) {
	hipMallocManaged(&delta, sizeof(int) * delta_size);
}

int getTransitionPos(TransitionRule *t) {
	int pos = t->to->controlLocation * (abpds_info->stack_size) + t->to->stack1;
	return pos;
}

void addRuleToDelta(TransitionRule *t) {
	//头插法
	int pos = getTransitionPos(t);
	if (delta[pos].next == NULL) {
		delta[pos].next = t;
	} else {
		t->next = delta[pos].next;
		delta[pos].next = t;
	}
}
__device__ void printRule(TransitionRule *rule) {
	if (rule != NULL) {
		if (rule->to_config_size == 1) {
			printf("<%d,%d>--><%d,%d %d>\n", rule->from.controlLocation,
					rule->from.stack, rule->to[0].controlLocation,
					rule->to[0].stack1, rule->to[0].stack2);
		}
	} else {
		printf("no rule\n");
	}
}

