#include "abpds.cuh"

Delta delta;

ABPDSInfo *abpds_info;

int *finalStateArray;
void initABPDSInfo() {
	hipMallocManaged(&abpds_info, sizeof(int) * 2);
}

void initDelta(int delta_size) {
	hipMallocManaged(&delta, sizeof(int) * delta_size);
}

int getTransitionPos(TransitionRule *t) {
	int pos = t->to->controlLocation * (abpds_info->stack_size) + t->to->stack1;
	return pos;
}

void addRuleToDelta(TransitionRule *t) {
	//头插法
	int pos=getTransitionPos(t);
	if(delta[pos].next==NULL)
	{
		delta[pos].next=t;
	}
	else{
		t->next=delta[pos].next;
		delta[pos].next=t;
	}
}


