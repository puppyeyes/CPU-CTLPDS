#include "hip/hip_runtime.h"
#include"ama.cuh"
using namespace std;
#define STATEMASK 0x3ff
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}
__device__ __host__ void insertTransToAMA(int amaListPosition, int state, AMA *ama, Pool *pool) {
	if (!isTransInAMA(t, ama, abpds_info)) {
		int pool_position = pool->tail++;
		if (pool_position > pool->size) {
			printf("pool exceed \n");
		}
		if (t.toState != -1) {
			ama->count++;
		}
		pool->item[pool_position].state = state;
		//尾插
		ama->list[amaListPosition].tail->next = &(pool->item[pool_position]);
		ama->list[amaListPosition].tail = &(pool->item[pool_position]);
		ama->list[amaListPosition].count++;
	}

}
__device__ void d_insertTransToAMA(int amaListPosition, int state, AMA *ama,
		Pool *pool, ABPDSInfo *abpds_info) {
	bool next = true;
	while (next) {
		int v = atomicCAS(&(ama->list[insertPosition].mutex), 0, 1);
		if (v == 0) {
			//在此放置你的临界区
			if (!isTransInAMA(t, ama, abpds_info)) {
				int pool_position = pool->tail++;
				if (pool_position > pool->size) {
					printf("pool exceed \n");
				}
				if (t.toState != -1) {
					ama->count++;
				}
				pool->item[pool_position].state = state;
				//尾插
				ama->list[amaListPosition].tail->next =
						&(pool->item[pool_position]);
				ama->list[amaListPosition].tail = &(pool->item[pool_position]);
				ama->list[amaListPosition].count++;

			}
			//临界区结束
			atomicExch(&(ama->list[amaListPosition].mutex), 0);
			next = false;
		}  //此处是安全的汇聚点
	}  //此处是安全的汇聚点2
}
void deleteAMA(AMA *ama, Pool *pool) {
	ama->count = 0;
	for (int i = 0; i < AMASIZE; i++) {
		ama->list[i].head.next = NULL;
	}
	pool->tail = 0;
}
bool isEqual(AMA *ama_1, AMA *ama_2) {
	if (ama_1->count == ama_2->count) {
		return true;
	}
	return false;
}
bool isFinalState(int state) {
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		if (state == finalStateArray[i]) {
			return true;
		}
	}
	return false;
}
__device__ __host__ bool isTransInAMA(Trans t, AMA *ama,
		ABPDSInfo *abpds_info) {
	int pos = t.fromState * abpds_info->stack_size + t.stack;
	AMANode *currentNode = ama->list[pos].head.next;
	while (currentNode != NULL) {
		if (currentNode->state == t.toState) {
			return true;
		}
		currentNode = currentNode->next;
	}
	return false;
}

__global__ void updateAMA(AMA *ama, int recursion, Pool *pool,
		ABPDSInfo *abpds_info) {
	//需要一个数组存储每个AMAList的所有元素  这个数组空间在哪开？
	int amaListPosition = threadIdx.x + blockIdx.x * blockDim.x;
	int *tmpStateList;
	int tmpListPosition = 0;
	hipMallocManaged(&tmpStateList,
			sizeof(int) * ama->list[amaListPosition].count);
	AMANode *currentNode = ama->list[amaListPosition].head.next;
	while (currentNode != NULL) {
		//updateState
		tmpStateList[tmpListPosition++] = encode_state_superScript(
				currentNode->state, recursion);
		currentNode = currentNode->next;
	}
	//断链
	ama->list[amaListPosition].head.next=NULL;
	ama->list[amaListPosition].tail=ama->list[amaListPosition].head;
	//重新插入
	for (int i = 0; i < tmpListPosition; i++) {
		insertTransToAMA(amaListPosition,tmpStateList[i],ama,pool,abpds_info);
	}
}

void initAMA(AMA *ama, Pool *pool) {
int amaSize = abpds_info->stack_size * abpds_info->state_size;
//	CUDA_CHECK_RETURN(hipMallocManaged(&ama, sizeof(AMA)));
CUDA_CHECK_RETURN(hipMallocManaged(&ama->list, sizeof(AMAList) * amaSize));
//初始化ama.list
for (int i = 0; i < amaSize; i++) {
	ama->list[i].mutex = 0;
	ama->list[i].count = 0;
	//尾指针指向头结点
	ama->list[i].tail = &(ama->list[i].head);
}
ama->count = amaSize;
//	CUDA_CHECK_RETURN(hipMallocManaged(&pool, sizeof(Pool)));
CUDA_CHECK_RETURN(
		hipMallocManaged (&pool->item, sizeof(AMANode) * AMAPOOLSIZE));
pool->size = AMAPOOLSIZE;
pool->tail = 0;
//finalStateArray 用-1表示
//		for(int i=0;i<abpds_info->finalStateSize; i++){
//			for(int j=0;j<abpds_info->stack_size;j++){
//				Trans t={finalStateArray[i],j,-1};
//				insertTransToAMA(t,ama,pool);
//			}
//		}
//cout<<isTransInAMA(t2,ama);
}
void printAMA(AMA *ama) {
map<int, string>::iterator it_find;
string from_state;
string stack;
string to_state;
cout << "打印结果" << endl;
for (int i = 0; i < (abpds_info->state_size); i++) {
	for (int j = 0; j < (abpds_info->stack_size); j++) {
		AMANode *tem_node = ama->list[i * abpds_info->stack_size + j].head.next;
		bool flag = false;
		if (tem_node != NULL) {
			flag = true;
			it_find = rv_state_mp.find(i);
			if (it_find != rv_state_mp.end()) {
				from_state = it_find->second;
			}
			it_find = rv_stack_mp.find(j);
			if (it_find != rv_state_mp.end()) {
				stack = it_find->second;
			}
			cout << from_state << " " << stack << "-->{ ";
		}
		while (tem_node != NULL) {
			short int superScript = decode_state_superScript(tem_node->state);
			it_find = rv_state_mp.find(tem_node->state & STATEMASK);
			if (it_find != rv_state_mp.end()) {
				to_state = it_find->second;
			} else {
				if (tem_node->state == -1) {
					to_state = "Qf";
				}
			}
			cout << to_state << ",";
			tem_node = tem_node->next;
		}
		if (flag) {
			cout << "}" << endl;
			flag = false;
		}
	}
}
cout << "结果输出结束" << endl;
}

