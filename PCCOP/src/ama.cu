#include "hip/hip_runtime.h"
#include"ama.cuh"
#include <stdio.h>
void insertTransToAMA(Trans t,AMA *ama,Pool *pool){
	int pool_position = pool->tail++;
	if (pool_position > pool->size){
			printf("pool exceed \n");
	}
	if(t.toState!=00000){
		ama->count++;
	}
	pool->item[pool_position].state= t.toState;
	int insertPosition=t.fromState*t.stack;
	//尾插
	ama->list[insertPosition].tail->next=&(pool->item[pool_position]);
	ama->list[insertPosition].tail=&(pool->item[pool_position]);
}
__device__ void d_insertTransToAMA(Trans t,AMA *ama,Pool *pool){
	int pool_position = atomicAdd(&(pool->tail), 1);
	if (pool_position > pool->size){
		printf("pool exceed \n");
	}
	if(t.toState!=00000){
		atomicAdd(&(ama->count), 1);
	}
	pool->item[pool_position].state= t.toState;
	int insertPosition=t.fromState*t.stack;
	bool next = true;
	while (next) {
		int v = atomicCAS(&(ama->list[insertPosition].mutex), 0, 1);
		if (v == 0) {
			//在此放置你的临界区
			//尾插
			ama->list[insertPosition].tail->next=&(pool->item[pool_position]);
			ama->list[insertPosition].tail=&(pool->item[pool_position]);
			//临界区结束
			atomicExch(&(ama->list[insertPosition].mutex), 0);
			next = false;
		}  //此处是安全的汇聚点
	}  //此处是安全的汇聚点2
}
void deleteAMA(AMA *ama,Pool *pool){
	ama->count=0;
	for (int i = 0; i < AMASIZE; i++) {
		ama->list[i].head.next=NULL;
	}
	pool->tail=0;
}
bool isEqual(AMA *ama_1,AMA *ama_2){
	if(ama_1->count==ama_2->count){
		return true;
	}
	return false;
}
bool isTransInAMA(Trans t,AMA *ama){

	AMANode *currentNode=ama->list[t.fromState*t.stack].head.next;
	while(currentNode!=NULL){
		if(currentNode->state==t.toState){
			return true;
		}
		currentNode=currentNode->next;
	}
	return false;
}
