#include "hip/hip_runtime.h"
#include"ama.cuh"

TMPAMA tmp_ama;
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}

void initTMP() {
	CUDA_SAFE_CALL(
			hipMallocManaged(&tmp_ama,
					sizeof(TMPINDEX) * abpds_info->state_size
							* abpds_info->stack_size));
}

void deleteTMP() {
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		for (int j = 0; j < abpds_info->stack_size; j++) {
			tmp_ama[finalStateArray[i] * abpds_info->stack_size + j].next =
			NULL;
		}
	}
}

void add_to_TMP(AMA *ama) {
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		for (int j = 0; j < abpds_info->stack_size; j++) {
			tmp_ama[finalStateArray[i] * abpds_info->stack_size + j].next =
					ama->list[finalStateArray[i] * abpds_info->stack_size + j].head.next;
		}
	}
}

bool insertTransToAMA(Trans t, AMA *ama, Pool *pool) {
	//if (!isTransInAMA(t, ama, abpds_info)) {
	int insertPosition = t.fromState * abpds_info->stack_size + t.stack;
	AMANode *currentNode = ama->list[insertPosition].head.next;
	if (currentNode == NULL) {
		//插入
		int pool_position = pool->tail++;
		if (pool_position > pool->size) {
			printf("pool exceed \n");
		}
		/*		if (t.toState != -1) {
		 ama->count++;
		 }*/
		pool->item[pool_position].state = t.toState;
		pool->item[pool_position].next = NULL;
		ama->list[insertPosition].tail->next = &(pool->item[pool_position]);
		ama->list[insertPosition].tail = &(pool->item[pool_position]);
		return true;
	}
	//head->3  插-1 head->-1->3
	if ((currentNode->state > t.toState) && currentNode != NULL) {
		//插入
		int pool_position = pool->tail++;
		if (pool_position > pool->size) {
			printf("pool exceed \n");
		}
		/*		if (t.toState != -1) {
		 ama->count++;
		 }*/
		pool->item[pool_position].state = t.toState;
		pool->item[pool_position].next = NULL;
		ama->list[insertPosition].head.next = &(pool->item[pool_position]);
		pool->item[pool_position].next = currentNode;
		return true;
	}
	while (currentNode != NULL) {
		if (currentNode->state < t.toState
				&& (currentNode->next == NULL
						|| currentNode->next->state > t.toState)) {
			//插入
			int pool_position = pool->tail++;
			if (pool_position > pool->size) {
				printf("pool exceed \n");
				return false;
			}
			/*			if (t.toState != -1) {
			 ama->count++;
			 }*/
			pool->item[pool_position].state = t.toState;
			pool->item[pool_position].next = NULL;
			AMANode *tmp = currentNode->next;
			currentNode->next = &(pool->item[pool_position]);
			pool->item[pool_position].next = tmp;
			return true;
		} else if (currentNode->state == t.toState) {
			return false;
		}
		currentNode = currentNode->next;
	}
	return false;
}

__device__ bool d_insertTransToAMA(Trans t, AMA *ama, Pool *pool,
		ABPDSInfo *abpds_info) {
	int insertPosition = t.fromState * abpds_info->stack_size + t.stack;
	//bool flag = false;
	bool next = true;

	while (next) {
		int v = atomicCAS(&(ama->list[insertPosition].mutex), 0, 1);
		if (v == 0) {
			//在此放置你的临界区
			AMANode *currentNode = ama->list[insertPosition].head.next;
			if (currentNode == NULL) {
				//插入
				int pool_position = atomicAdd(&pool->tail, 1);
				if (pool_position > pool->size) {
					printf("pool exceed \n");
				}
				/*				if (t.toState != -1) {
				 atomicAdd( &ama->count,1);
				 }*/
				pool->item[pool_position].state = t.toState;
				pool->item[pool_position].next = NULL;

				ama->list[insertPosition].tail->next =
						&(pool->item[pool_position]);
				ama->list[insertPosition].tail = &(pool->item[pool_position]);
				atomicExch(&(ama->list[insertPosition].mutex), 0);
				return true;
			}
			//head->3  插-1 head->-1->3
			if (currentNode != NULL && (currentNode->state > t.toState)) {
				//插入
				int pool_position = atomicAdd(&pool->tail, 1);
				if (pool_position > pool->size) {
					printf("pool exceed \n");
				}
				/*				if (t.toState != -1) {
				 atomicAdd( &ama->count,1);
				 }*/
				pool->item[pool_position].state = t.toState;
				pool->item[pool_position].next = NULL;
				ama->list[insertPosition].head.next =
						&(pool->item[pool_position]);
				pool->item[pool_position].next = currentNode;
				atomicExch(&(ama->list[insertPosition].mutex), 0);
				return true;
			}
			while (currentNode != NULL) {
				if (currentNode->state < t.toState
						&& (currentNode->next == NULL
								|| currentNode->next->state > t.toState)) {
					//插入
					int pool_position = atomicAdd(&pool->tail, 1);
					if (pool_position > pool->size) {
						printf("ama->list[i].headpool exceed \n");
						atomicExch(&(ama->list[insertPosition].mutex), 0);
						return false;
					}
					/*					if (t.toState != -1) {
					 atomicAdd( &ama->count,1);
					 }*/
					pool->item[pool_position].state = t.toState;
					pool->item[pool_position].next = NULL;
					AMANode *tmp = currentNode->next;
					currentNode->next = &(pool->item[pool_position]);
					pool->item[pool_position].next = tmp;
					atomicExch(&(ama->list[insertPosition].mutex), 0);
					return true;
				} else if (currentNode->state == t.toState) {
					atomicExch(&(ama->list[insertPosition].mutex), 0);
					return false;
				}
				currentNode = currentNode->next;
			}
			//临界区结束
			atomicExch(&(ama->list[insertPosition].mutex), 0);
			next = false;
		}  //此处是安全的汇聚点
	}  //此处是安全的汇聚点2
	return false;
}

__device__ bool d_insertStateToAMA(int insertPosition, int state, AMA *ama,
		Pool *pool) {
	AMANode *currentNode = ama->list[insertPosition].head.next;
//	bool flag = false;
	if (currentNode == NULL) {
		//插入
		int pool_position = atomicAdd(&pool->tail, 1);
		if (pool_position > pool->size) {
			printf("pool exceed \n");
		}
		if (state != -1) {
			atomicAdd(&ama->count, 1);
		}
		pool->item[pool_position].state = state;
		pool->item[pool_position].next = NULL;
		ama->list[insertPosition].tail->next = &(pool->item[pool_position]);
		ama->list[insertPosition].tail = &(pool->item[pool_position]);
		return true;
	}
	//head->3  插-1 head->-1->3
	if (currentNode != NULL && (currentNode->state > state)) {
		//插入
		int pool_position = atomicAdd(&pool->tail, 1);
		if (pool_position > pool->size) {
			printf("pool exceed \n");
		}
		if (state != -1) {
			atomicAdd(&ama->count, 1);
		}
		pool->item[pool_position].state = state;
		pool->item[pool_position].next = NULL;
		ama->list[insertPosition].head.next = &(pool->item[pool_position]);
		pool->item[pool_position].next = currentNode;
		return true;
	}
	while (currentNode != NULL) {
		if (currentNode->state < state
				&& (currentNode->next == NULL
						|| currentNode->next->state > state)) {
			//插入
			int pool_position = atomicAdd(&pool->tail, 1);
			if (pool_position > pool->size) {
				printf("pool exceed \n");
				return false;
			}
			if (state != -1) {
				atomicAdd(&ama->count, 1);
			}
			pool->item[pool_position].state = state;
			pool->item[pool_position].next = NULL;
			AMANode *tmp = currentNode->next;
			currentNode->next = &(pool->item[pool_position]);
			pool->item[pool_position].next = tmp;
			return true;
		} else if (currentNode->state == state) {
			//printf("--id %d-- %d  exist\n",insertPosition,currentNode->state);
			return false;
		}
		currentNode = currentNode->next;
	}
	return false;

//	int pool_position = pool->tail++;
//	if (pool_position > pool->size) {
//		printf("pool exceed \n");
//	}
//	pool->item[pool_position].state = state;
//	//尾插
//	ama->list[amaListPosition].tail->next = &(pooupdateAMAl->item[pool_position]);
//	ama->list[amaListPosition].tail = &(pool->item[pool_position]);

}
void deleteAMA(AMA *ama, Pool *pool) {
	ama->count = 0;
	for (int i = 0; i < abpds_info->state_size * abpds_info->stack_size; i++) {
		ama->list[i].head.next = NULL;
		//尾指针指向头结点
		ama->list[i].tail = &(ama->list[i].head);
		ama->list[i].mutex = 0;
		ama->list[i].count = 0;
	}
	pool->tail = 0;

}
bool isEqual(AMA *ama_1, AMA *ama_2) {
	if (ama_1->count == ama_2->count) {
		return true;
	}
	return false;
}
bool isFinalState(int state) {
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		if (state == finalStateArray[i]) {
			return true;
		}
	}
	return false;
}
__device__ __host__ bool isTransInAMA(Trans t, AMA *ama,
		ABPDSInfo *abpds_info) {
	int pos = t.fromState * abpds_info->stack_size + t.stack;
	AMANode *currentNode = ama->list[pos].head.next;
	while (currentNode != NULL) {
		if (currentNode->state == t.toState) {
			return true;
		}
		currentNode = currentNode->next;
	}
	return false;
}

__global__ void updateAMA(AMA *ama, int recursion, Pool *pool,
		ABPDSInfo *abpds_info) {
	//需要一个数组存储每个AMAList的所有元素  这个数组空间在哪开？
	int amaListPosition = threadIdx.x + blockIdx.x * blockDim.x;
	//int *tmpStateList;
	if (amaListPosition < abpds_info->stack_size * abpds_info->state_size
			&& (amaListPosition == 7 | 17)) {
		int tmpListPosition = 0;
		int *tmpStateList;
		tmpStateList = (int *) malloc(
				sizeof(int) * ama->list[amaListPosition].count);
//	hipMallocManaged(&tmpStateList,
//			sizeof(int) * ama->list[amaListPosition].count);
		AMANode *currentNode = ama->list[amaListPosition].head.next;
		while (currentNode != NULL) {
			//updateState
			if (currentNode->state != -1) {
				tmpStateList[tmpListPosition++] = encode_state_superScript(
						currentNode->state, recursion);
			} else {
				tmpStateList[tmpListPosition++] = currentNode->state;
			}
			currentNode = currentNode->next;
		}
		//断链
		ama->list[amaListPosition].head.next = NULL;
		ama->list[amaListPosition].tail = &(ama->list[amaListPosition].head);
		ama->list[amaListPosition].count = 0;
		//重新插入
		for (int i = 0; i < tmpListPosition; i++) {
			int result = d_insertStateToAMA(amaListPosition, tmpStateList[i],
					ama, pool);
		}
	}

}

void initAMA(AMA *ama, Pool *pool) {
	int amaSize = abpds_info->stack_size * abpds_info->state_size;
//	CUDA_CHECK_RETURN(hipMallocManaged(&ama, sizeof(AMA)));
	CUDA_CHECK_RETURN(hipMallocManaged(&ama->list, sizeof(AMAList) * amaSize));
//初始化ama.list
	for (int i = 0; i < amaSize; i++) {
		ama->list[i].mutex = 0;
		ama->list[i].count = 0;
		//尾指针指向头结点
		ama->list[i].tail = &(ama->list[i].head);
	}
	ama->count = 0;
//	CUDA_CHECK_RETURN(hipMallocManaged(&pool, sizeof(Pool)));
	CUDA_CHECK_RETURN(
			hipMallocManaged (&pool->item, sizeof(AMANode) * AMAPOOLSIZE));
	pool->size = AMAPOOLSIZE;
	pool->tail = 0;
//finalStateArray 用-1表示
//		for(int i=0;i<abpds_info->finalStateSize; i++){
//			for(int j=0;j<abpds_info->stack_size;j++){
//				Trans t={finalStateArray[i],j,-1};
//				insertTransToAMA(t,ama,pool);
//			}
//		}
//cout<<isTransInAMA(t2,ama);
}

void printAMA(AMA *ama) {
	map<int, string>::iterator it_find;
	string from_state;
	string stack;
	string to_state;
	string super_script;
	int state_id;
	int stack_id;
	cout << "打印结果" << endl;
	for (int i = 0; i < (abpds_info->state_size); i++) {
		for (int j = 0; j < (abpds_info->stack_size); j++) {
			AMANode *tem_node =
					ama->list[i * abpds_info->stack_size + j].head.next;
			bool flag = false;
			if (tem_node != NULL) {
				flag = true;
				it_find = rv_state_mp.find(i);
				if (it_find != rv_state_mp.end()) {
					from_state = it_find->second;
					state_id = i;
				}
				it_find = rv_stack_mp.find(j);
				if (it_find != rv_state_mp.end()) {
					stack = it_find->second;
					stack_id = j;
				}
				cout << from_state;
				//printf("(%d)", state_id);
				cout << " " << stack;
				//printf("(%d)", stack_id);
				cout << "-->{ ";
			}
			while (tem_node != NULL) {
				short int superScript = decode_state_superScript(
						tem_node->state);
				it_find = rv_state_mp.find(tem_node->state & STATEMASK);
				if (it_find != rv_state_mp.end()) {
					to_state = it_find->second;
				} else {
					if (tem_node->state == -1) {
						to_state = "Qf";
					}
				}
				cout << to_state;
				if (tem_node->state != -1) {
					printf("[%d]",
							(int) ((tem_node->state & SPUPERSCRIPTMASK)
									>> STATEBIT));
				}
				//cout << "("<<tem_node->state<<")"<<",";
				cout << ",";
				tem_node = tem_node->next;
			}
			if (flag) {
				cout << "}" << endl;
				flag = false;
			}
		}
	}
	cout << "结果输出结束" << endl;
}
