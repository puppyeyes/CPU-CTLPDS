#include "hip/hip_runtime.h"
#include"ama.cuh"
using namespace std;
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}
void insertTransToAMA(Trans t, AMA *ama, Pool *pool) {
	int pool_position = pool->tail++;
	if (pool_position > pool->size) {
		printf("pool exceed \n");
	}
	if (t.toState != 00000) {
		ama->count++;
	}
	pool->item[pool_position].state = t.toState;
	int insertPosition = t.fromState *abpds_info->stack_size+ t.stack;
	//尾插
	ama->list[insertPosition].tail->next = &(pool->item[pool_position]);
	ama->list[insertPosition].tail = &(pool->item[pool_position]);
}
__device__ void d_insertTransToAMA(Trans t, AMA *ama, Pool *pool,ABPDSInfo *abpds_info) {
	int pool_position = atomicAdd(&(pool->tail), 1);
	if (pool_position > pool->size) {
		printf("pool exceed \n");
	}
	if (t.toState != 00000) {
		atomicAdd(&(ama->count), 1);
	}
	pool->item[pool_position].state = t.toState;
	int insertPosition = t.fromState * abpds_info->stack_size+t.stack;
	bool next = true;
	while (next) {
		int v = atomicCAS(&(ama->list[insertPosition].mutex), 0, 1);
		if (v == 0) {
			//在此放置你的临界区
			//尾插
			ama->list[insertPosition].tail->next = &(pool->item[pool_position]);
			ama->list[insertPosition].tail = &(pool->item[pool_position]);
			//临界区结束
			atomicExch(&(ama->list[insertPosition].mutex), 0);
			next = false;
		}  //此处是安全的汇聚点
	}  //此处是安全的汇聚点2
}
void deleteAMA(AMA *ama, Pool *pool) {
	ama->count = 0;
	for (int i = 0; i < AMASIZE; i++) {
		ama->list[i].head.next = NULL;
	}
	pool->tail = 0;
}
bool isEqual(AMA *ama_1, AMA *ama_2) {
	if (ama_1->count == ama_2->count) {
		return true;
	}
	return false;
}
__device__ __host__ bool isTransInAMA(Trans t, AMA *ama,ABPDSInfo *abpds_info) {
	int pos=t.fromState*abpds_info->stack_size+ t.stack;
	AMANode *currentNode = ama->list[pos].head.next;
	while (currentNode != NULL) {
		if (currentNode->state == t.toState) {
			return true;
		}
		currentNode = currentNode->next;
	}
	return false;
}

void initAMA(AMA *ama, Pool *pool) {
	int amaSize = abpds_info->stack_size * abpds_info->state_size;
//	CUDA_CHECK_RETURN(hipMallocManaged(&ama, sizeof(AMA)));
	CUDA_CHECK_RETURN(hipMallocManaged(&ama->list, sizeof(AMAList) * amaSize));
	//初始化ama.list
	for (int i = 0; i < amaSize; i++) {
		ama->list[i].mutex = 0;
		//尾指针指向头结点
		ama->list[i].tail = &(ama->list[i].head);
	}
	ama->count=amaSize;
//	CUDA_CHECK_RETURN(hipMallocManaged(&pool, sizeof(Pool)));
	CUDA_CHECK_RETURN(
			hipMallocManaged (&pool->item, sizeof(AMANode) * AMAPOOLSIZE));
	pool->size = AMAPOOLSIZE;
	pool->tail = 0;
	//finalStateArray 用-1表示
//		for(int i=0;i<abpds_info->finalStateSize; i++){
//			for(int j=0;j<abpds_info->stack_size;j++){
//				Trans t={finalStateArray[i],j,-1};
//				insertTransToAMA(t,ama,pool);
//			}
//		}
	//cout<<isTransInAMA(t2,ama);
}
void printAMA(AMA *ama) {
	map<int, string>::iterator it_find;
	string from_state;
	string stack;
	string to_state;
	cout <<"打印结果"<<endl;
	for (int i = 0; i < (abpds_info->state_size); i++) {
		for (int j = 0; j < (abpds_info->stack_size); j++) {
			AMANode *tem_node =
					ama->list[i * abpds_info->stack_size + j].head.next;
			while (tem_node != NULL) {
				it_find = rv_state_mp.find(i);
				if (it_find != rv_state_mp.end()) {
					from_state = it_find->second;
				}
				it_find = rv_stack_mp.find(j);
				if (it_find != rv_state_mp.end()) {
					stack = it_find->second;
				}
				it_find = rv_state_mp.find(tem_node->state);
				if (it_find != rv_state_mp.end()) {
					to_state = it_find->second;
				}else{
					if(tem_node->state==-1)
					{
						to_state="Qf";
					}
				}
				cout << from_state <<" "<< stack << "-->" << to_state << endl;
				tem_node = tem_node->next;
			}

		}
	}
	cout <<"结果输出结束"<<endl;
}

