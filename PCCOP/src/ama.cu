#include "hip/hip_runtime.h"
#include"ama.cuh"
#include <stdio.h>
void insertTransToAMA(Trans t,AMA *ama,Pool *pool){
	int pool_position = pool->tail++;
	if (pool_position > pool->size){
			printf("pool exceed \n");
	}
	if(t.toState!=00000){
		ama->count++;
	}
	pool->item[pool_position].state= t.toState;
	int insertPosition=t.fromState*t.stack;
	//尾插
	ama->list[insertPosition].tail->next=&(pool->item[pool_position]);
	ama->list[insertPosition].tail=&(pool->item[pool_position]);
}
__device__ void d_insertTransToAMA(Trans t,AMA *ama,Pool *pool){
	int pool_position = atomicAdd(&(pool->tail), 1);
	if (pool_position > pool->size){
		printf("pool exceed \n");
	}
	if(t.toState!=00000){
		atomicAdd(&(ama->count), 1);
	}
	pool->item[pool_position].state= t.toState;
	int insertPosition=t.fromState*t.stack;
	bool next = true;
	while (next) {
		int v = atomicCAS(&(ama->list[insertPosition].mutex), 0, 1);
		if (v == 0) {
			//在此放置你的临界区
			//尾插
			ama->list[insertPosition].tail->next=&(pool->item[pool_position]);
			ama->list[insertPosition].tail=&(pool->item[pool_position]);
			//临界区结束
			atomicExch(&(ama->list[insertPosition].mutex), 0);
			next = false;
		}  //此处是安全的汇聚点
	}  //此处是安全的汇聚点2
}
void deleteAMA(AMA *ama,Pool *pool){
	ama->count=0;
	for (int i = 0; i < AMASIZE; i++) {
		ama->list[i].head.next=NULL;
	}
	pool->tail=0;
}
bool isEqual(AMA *ama_1,AMA *ama_2){
	if(ama_1->count==ama_2->count){
		return true;
	}
	return false;
}
__device__ __host__ bool isTransInAMA(Trans t,AMA *ama){

	AMANode *currentNode=ama->list[t.fromState*t.stack].head.next;
	while(currentNode!=NULL){
		if(currentNode->state==t.toState){
			return true;
		}
		currentNode=currentNode->next;
	}
	return false;
}

/*void initAMA(AMA *ama,Pool *pool){
		int amaSize=abpds_info->stack_size*abpds_info->state_size;
		CUDA_CHECK_RETURN(hipMallocManaged(&ama, sizeof(AMA)));
		CUDA_CHECK_RETURN(hipMallocManaged (&ama->list, sizeof(AMAList)*amaSize));
		//初始化ama.list
		for (int i = 0; i < amaSize; i++) {
			ama->list[i].mutex = 0;
			//尾指针指向头结点
			ama->list[i].tail=&(ama->list[i].head);
		}
		CUDA_CHECK_RETURN(hipMallocManaged(&pool, sizeof(Pool)));
		CUDA_CHECK_RETURN(hipMallocManaged (&pool->item, sizeof(AMANode) * AMAPOOLSIZE));
		pool->size=AMAPOOLSIZE;
		pool->tail=0;
		//finalStateArray 用-1表示
//		for(int i=0;i<abpds_info->finalStateSize; i++){
//			for(int j=0;j<abpds_info->stack_size;j++){
//				Trans t={finalStateArray[i],j,-1};
//				insertTransToAMA(t,ama,pool);
//			}
//		}
		//cout<<isTransInAMA(t2,ama);
}*/
