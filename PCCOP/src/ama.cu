#include "hip/hip_runtime.h"
#include"ama.cuh"
using namespace std;
#define STATEMASK 0x3ff
#define STATEBIT 10
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}
bool insertTransToAMA(Trans t, AMA *ama, Pool *pool) {
	//if (!isTransInAMA(t, ama, abpds_info)) {
	int insertPosition = t.fromState * abpds_info->stack_size + t.stack;
	AMANode *currentNode = ama->list[insertPosition].head.next;
	if (currentNode == NULL) {
		//插入
		int pool_position = pool->tail++;
		if (pool_position > pool->size) {
			printf("pool exceed \n");
		}
		if (t.toState != -1) {
			ama->count++;
		}
		pool->item[pool_position].state = t.toState;

		ama->list[insertPosition].tail->next = &(pool->item[pool_position]);
		ama->list[insertPosition].tail = &(pool->item[pool_position]);
		return true;
	}
	while (currentNode != NULL) {
		if (currentNode->state < t.toState
				&& (currentNode->next == NULL
						|| currentNode->next->state > t.toState)) {
			//插入
			int pool_position = pool->tail++;
			if (pool_position > pool->size) {
				printf("pool exceed \n");
				return false;
			}
			if (t.toState != -1) {
				ama->count++;
			}
			pool->item[pool_position].state = t.toState;
			AMANode *tmp = currentNode->next;
			currentNode->next = &(pool->item[pool_position]);
			pool->item[pool_position].next = tmp;
			return true;
		} else if (currentNode->state == t.toState) {
			return false;
		}
		currentNode = currentNode->next;
	}
	return false;
}
__device__ bool d_insertTransToAMA(Trans t, AMA *ama, Pool *pool,
		ABPDSInfo *abpds_info) {

	int insertPosition = t.fromState * abpds_info->stack_size + t.stack;
	bool flag=false;
	bool next = true;
	while (next) {
		int v = atomicCAS(&(ama->list[insertPosition].mutex), 0, 1);
		if (v == 0) {
			//在此放置你的临界区
			AMANode *currentNode = ama->list[insertPosition].head.next;
			if (currentNode == NULL) {
				//插入
				int pool_position = pool->tail++;
				if (pool_position > pool->size) {
					printf("pool exceed \n");
				}
				if (t.toState != -1) {
					ama->count++;
				}
				pool->item[pool_position].state = t.toState;

				ama->list[insertPosition].tail->next =
						&(pool->item[pool_position]);
				ama->list[insertPosition].tail = &(pool->item[pool_position]);
				flag= true;
			}
			while (currentNode != NULL) {
				if (currentNode->state < t.toState
						&& (currentNode->next == NULL
								|| currentNode->next->state > t.toState)) {
					//插入
					int pool_position = pool->tail++;
					if (pool_position > pool->size) {
						printf("pool exceed \n");
						flag= false;
					}
					if (t.toState != -1) {
						ama->count++;
					}
					pool->item[pool_position].state = t.toState;
					AMANode *tmp = currentNode->next;
					currentNode->next = &(pool->item[pool_position]);
					pool->item[pool_position].next = tmp;
					flag= true;
				} else if (currentNode->state == t.toState) {
					flag= false;
				}
				currentNode = currentNode->next;
			}
			//临界区结束
			atomicExch(&(ama->list[insertPosition].mutex), 0);
			next = false;
		}  //此处是安全的汇聚点
	}  //此处是安全的汇聚点2
	return flag;
}

__device__ bool d_insertStateToAMA(int insertPosition, int state, AMA *ama,
		Pool *pool) {
		AMANode *currentNode = ama->list[insertPosition].head.next;
		if (currentNode == NULL) {
			//插入
			int pool_position = pool->tail++;
			if (pool_position > pool->size) {
				printf("pool exceed \n");
			}
			if (state != -1) {
				ama->count++;
			}
			pool->item[pool_position].state =state;
			ama->list[insertPosition].tail->next = &(pool->item[pool_position]);
			ama->list[insertPosition].tail = &(pool->item[pool_position]);
			return true;
		}
		while (currentNode != NULL) {
			if (currentNode->state < state
					&& (currentNode->next == NULL
							|| currentNode->next->state > state)) {
				//插入
				int pool_position = pool->tail++;
				if (pool_position > pool->size) {
					printf("pool exceed \n");
					return false;
				}
				if (state != -1) {
					ama->count++;
				}
				pool->item[pool_position].state = state;
				AMANode *tmp = currentNode->next;
				currentNode->next = &(pool->item[pool_position]);
				pool->item[pool_position].next = tmp;
				return true;
			} else if (currentNode->state == state) {
				return false;
			}
			currentNode = currentNode->next;
		}
		return false;


//	int pool_position = pool->tail++;
//	if (pool_position > pool->size) {
//		printf("pool exceed \n");
//	}
//	pool->item[pool_position].state = state;
//	//尾插
//	ama->list[amaListPosition].tail->next = &(pool->item[pool_position]);
//	ama->list[amaListPosition].tail = &(pool->item[pool_position]);

}
void deleteAMA(AMA *ama, Pool *pool) {
	ama->count = 0;
	for (int i = 0; i < AMASIZE; i++) {
		ama->list[i].head.next = NULL;
	}
	pool->tail = 0;
}
bool isEqual(AMA *ama_1, AMA *ama_2) {
	if (ama_1->count == ama_2->count) {
		return true;
	}
	return false;
}
bool isFinalState(int state) {
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		if (state == finalStateArray[i]) {
			return true;
		}
	}
	return false;
}
__device__ __host__ bool isTransInAMA(Trans t, AMA *ama,
		ABPDSInfo *abpds_info) {
	int pos = t.fromState * abpds_info->stack_size + t.stack;
	AMANode *currentNode = ama->list[pos].head.next;
	while (currentNode != NULL) {
		if (currentNode->state == t.toState) {
			return true;
		}
		currentNode = currentNode->next;
	}
	return false;
}

__global__ void updateAMA(int *tmpStateList, AMA *ama, int recursion,
		Pool *pool, ABPDSInfo *abpds_info) {
	//需要一个数组存储每个AMAList的所有元素  这个数组空间在哪开？
	int amaListPosition = threadIdx.x + blockIdx.x * blockDim.x;
	//int *tmpStateList;
	int tmpListPosition = 0;
//	hipMallocManaged(&tmpStateList,
//			sizeof(int) * ama->list[amaListPosition].count);
	AMANode *currentNode = ama->list[amaListPosition].head.next;
	while (currentNode != NULL) {
		//updateState
		if (currentNode->state != -1) {
			tmpStateList[tmpListPosition++] = encode_state_superScript(
					currentNode->state, recursion);
			currentNode = currentNode->next;
		}
	}
	//断链
	ama->list[amaListPosition].head.next = NULL;
	ama->list[amaListPosition].tail = &(ama->list[amaListPosition].head);
	//重新插入
	for (int i = 0; i < tmpListPosition; i++) {
		d_insertStateToAMA(amaListPosition, tmpStateList[i], ama, pool);
	}
}

void initAMA(AMA *ama, Pool *pool) {
	int amaSize = abpds_info->stack_size * abpds_info->state_size;
//	CUDA_CHECK_RETURN(hipMallocManaged(&ama, sizeof(AMA)));
	CUDA_CHECK_RETURN(hipMallocManaged(&ama->list, sizeof(AMAList) * amaSize));
//初始化ama.list
	for (int i = 0; i < amaSize; i++) {
		ama->list[i].mutex = 0;
		ama->list[i].count = 0;
		//尾指针指向头结点
		ama->list[i].tail = &(ama->list[i].head);
	}
	ama->count = amaSize;
//	CUDA_CHECK_RETURN(hipMallocManaged(&pool, sizeof(Pool)));
	CUDA_CHECK_RETURN(
			hipMallocManaged (&pool->item, sizeof(AMANode) * AMAPOOLSIZE));
	pool->size = AMAPOOLSIZE;
	pool->tail = 0;
//finalStateArray 用-1表示
//		for(int i=0;i<abpds_info->finalStateSize; i++){
//			for(int j=0;j<abpds_info->stack_size;j++){
//				Trans t={finalStateArray[i],j,-1};
//				insertTransToAMA(t,ama,pool);
//			}
//		}
//cout<<isTransInAMA(t2,ama);
}

__device__ int encode_state_superScript(int state, short int recursion) {
	int res = state | (recursion << STATEBIT);
	return res;
}

__host__ __device__ short int decode_state_superScript(int state) {
	int res = state >> STATEBIT;
	return res;
}

void printAMA(AMA *ama) {
	map<int, string>::iterator it_find;
	string from_state;
	string stack;
	string to_state;
	cout << "打印结果" << endl;
	for (int i = 0; i < (abpds_info->state_size); i++) {
		for (int j = 0; j < (abpds_info->stack_size); j++) {
			AMANode *tem_node =
					ama->list[i * abpds_info->stack_size + j].head.next;
			bool flag = false;
			if (tem_node != NULL) {
				flag = true;
				it_find = rv_state_mp.find(i);
				if (it_find != rv_state_mp.end()) {
					from_state = it_find->second;
				}
				it_find = rv_stack_mp.find(j);
				if (it_find != rv_state_mp.end()) {
					stack = it_find->second;
				}
				cout << from_state << " " << stack << "-->{ ";
			}
			while (tem_node != NULL) {
				short int superScript = decode_state_superScript(
						tem_node->state);
				it_find = rv_state_mp.find(tem_node->state & STATEMASK);
				if (it_find != rv_state_mp.end()) {
					to_state = it_find->second;
				} else {
					if (tem_node->state == -1) {
						to_state = "Qf";
					}
				}
				cout << to_state << ",";
				tem_node = tem_node->next;
			}
			if (flag) {
				cout << "}" << endl;
				flag = false;
			}
		}
	}
	cout << "结果输出结束" << endl;
}
