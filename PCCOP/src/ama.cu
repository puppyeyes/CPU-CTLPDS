#include "hip/hip_runtime.h"
#include"ama.cuh"

TMPAMA tmp_ama;
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}

void initTMP() {
	//存放临时的边，数组大小为abpds_info->state_size* abpds_info->stack_size
	CUDA_SAFE_CALL(
			hipMallocManaged(&tmp_ama,
					sizeof(TMPINDEX) * abpds_info->state_size
							* abpds_info->stack_size));
}

void deleteTMP() {
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		for (int j = 0; j < abpds_info->stack_size; j++) {
			tmp_ama[finalStateArray[i] * abpds_info->stack_size + j].next =
			NULL;
			tmp_ama[finalStateArray[i] * abpds_info->stack_size + j].nodeFlag =
					0;
		}
	}
}

void add_to_TMP(AMA *ama) {
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		for (int j = 0; j < abpds_info->stack_size; j++) {
			tmp_ama[finalStateArray[i] * abpds_info->stack_size + j].nodeFlag =
					ama->list[finalStateArray[i] * abpds_info->stack_size + j].nodeFlag;
			tmp_ama[finalStateArray[i] * abpds_info->stack_size + j].next =
					ama->list[finalStateArray[i] * abpds_info->stack_size + j].next;
		}
	}
}

bool insertTransToAMA(Trans t, AMA *ama, Pool *pool) {
	//if (!isTransInAMA(t, ama, abpds_info)) {
	int insertPosition = t.fromState * abpds_info->stack_size + t.stack;
	if (!isExist(ama->list[insertPosition].nodeFlag, t.toState)) {
		//插入
		int pool_position = pool->tail++;
		if (pool_position > pool->size) {
			printf("pool exceed \n");
		}
		pool->item[pool_position].state = t.toState;
		pool->item[pool_position].next = NULL;
		pool->item[pool_position].next = ama->list[insertPosition].next;
		ama->list[insertPosition].next = &(pool->item[pool_position]);
		ama->list[insertPosition].nodeFlag |= (1
				<< ((t.toState & STATEMASK) + 1));
		return true;
	}
	return false;
}

__device__ __host__ bool isExist(unsigned long long int nodeFlag,
		short int state) {
	int res = 0;
	if (state == -1) {
		res = (nodeFlag) & 0x01;
	} else {
		res = (nodeFlag >> ((state & STATEMASK) + 1)) & 0x01;
	}
	if (res == 0) {
		return false;
	} else {
		return true;
	}
}

__device__ bool d_insertTransToAMA(Trans t, AMA *ama, Pool *pool,
		ABPDSInfo *abpds_info) {
	int insertPosition = t.fromState * abpds_info->stack_size + t.stack;
	bool next = true;
	unsigned long long int nodeFlag;
	while (next) {
		int v = atomicCAS(&(ama->list[insertPosition].mutex), 0, 1);
		if (v == 0) {
			//在此放置你的临界区
			if (!isExist(ama->list[insertPosition].nodeFlag, t.toState)) {
				if((nodeFlag) & 0x01){
					nodeFlag = ama->list[insertPosition].nodeFlag
												| (1 << ((t.toState & STATEMASK) + 1));
					atomicExch(&(ama->list[insertPosition].nodeFlag), nodeFlag);
					atomicExch(&(ama->list[insertPosition].mutex), 0);
					return false;
				}
				//插入
				int pool_position = atomicAdd(&(pool->tail), 1);
				if (pool_position > pool->size) {
					printf("pool exceed \n");
				}
				pool->item[pool_position].state = t.toState;
				pool->item[pool_position].next = NULL;
				pool->item[pool_position].next = ama->list[insertPosition].next;
				ama->list[insertPosition].next = &(pool->item[pool_position]);
				if (t.toState != -1) {
					nodeFlag = ama->list[insertPosition].nodeFlag
							| (1 << ((t.toState & STATEMASK) + 1));
				} else {
					nodeFlag = ama->list[insertPosition].nodeFlag | 1;
				}
				atomicExch(&(ama->list[insertPosition].nodeFlag), nodeFlag);
				atomicExch(&(ama->list[insertPosition].mutex), 0);
				return true;
			} else {
				atomicExch(&(ama->list[insertPosition].mutex),0);
				return false;
			}
			//临界区结束
		}  //此处是安全的汇聚点
	}  //此处是安全的汇聚点2
	return false;
}

void deleteAMA(AMA *ama, Pool *pool) {
	ama->count = 0;
	for (int i = 0; i < abpds_info->state_size * abpds_info->stack_size; i++) {
		//尾指针指向头结点
		ama->list[i].next = NULL;
		ama->list[i].mutex = 0;
		ama->list[i].nodeFlag = 0;
	}
	pool->tail = 0;

}
bool isEqual(AMA *ama_1, AMA *ama_2) {
	if (ama_1->count == ama_2->count) {
		return true;
	}
	return false;
}
bool isFinalState(int state) {
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		if (state == finalStateArray[i]) {
			return true;
		}
	}
	return false;
}

__global__ void updateAMA(AMA *ama, int recursion, Pool *pool,
		ABPDSInfo *abpds_info) {
//需要一个数组存储每个AMAList的所有元素  这个数组空间在哪开？
	int amaListPosition = threadIdx.x + blockIdx.x * blockDim.x;
//int *tmpStateList;
	if (amaListPosition < abpds_info->stack_size * abpds_info->state_size) {
		AMANode *currentNode = ama->list[amaListPosition].next;
		if(ama->list[amaListPosition].nodeFlag&0x01){
			currentNode->state=-1;
			ama->list[amaListPosition].next = currentNode;
				currentNode->next = NULL;
		}
		while (currentNode != NULL) {
			//updateState
			if (currentNode->state != -1) {
				currentNode->state = encode_state_superScript(
						currentNode->state, recursion);
				atomicAdd(&(ama->count),1);
			}
			currentNode = currentNode->next;
		}
	}
}

void initAMA(AMA *ama, Pool *pool) {
	int amaSize = abpds_info->stack_size * abpds_info->state_size;
//	CUDA_CHECK_RETURN(hipMallocManaged(&ama, sizeof(AMA)));
	CUDA_CHECK_RETURN(hipMallocManaged(&ama->list, sizeof(AMAList) * amaSize));
//初始化ama.list
	for (int i = 0; i < amaSize; i++) {
		ama->list[i].mutex = 0;
		ama->list[i].nodeFlag = 0;
		//尾指针指向头结点
		ama->list[i].next = NULL;
	}
	ama->count = 0;
//	CUDA_CHECK_RETURN(hipMallocManaged(&pool, sizeof(Pool)));
	CUDA_CHECK_RETURN(
			hipMallocManaged (&pool->item, sizeof(AMANode) * AMAPOOLSIZE));
	pool->size = AMAPOOLSIZE;
	pool->tail = 0;
}

void printAMA(AMA *ama) {
	map<int, string>::iterator it_find;
	string from_state;
	string stack;
	string to_state;
	string super_script;
	int state_id;
	int stack_id;
//int count=0;
	cout << "打印结果" << endl;
	for (int i = 0; i < (abpds_info->state_size); i++) {
		for (int j = 0; j < (abpds_info->stack_size); j++) {
			AMANode *tem_node = ama->list[i * abpds_info->stack_size + j].next;
			bool flag = false;
			if (tem_node != NULL) {
				flag = true;
				it_find = rv_state_mp.find(i);
				if (it_find != rv_state_mp.end()) {
					from_state = it_find->second;
					state_id = i;
				}
				it_find = rv_stack_mp.find(j);
				if (it_find != rv_state_mp.end()) {
					stack = it_find->second;
					stack_id = j;
				}
				cout << from_state;
				//printf("(%d)", state_id);
				cout << " " << stack;
				//printf("(%d)", stack_id);
				cout << "-->{ ";
			}
			while (tem_node != NULL) {
				short int superScript = decode_state_superScript(
						tem_node->state);
				it_find = rv_state_mp.find(tem_node->state & STATEMASK);
				if (it_find != rv_state_mp.end()) {
					to_state = it_find->second;
					//count++;
				} else {
					if (tem_node->state == -1) {
						to_state = "Qf";
					}
				}
				cout << to_state;
				if (tem_node->state != -1) {
					printf("[%d]",
							(int) ((tem_node->state & SPUPERSCRIPTMASK)
									>> STATEBIT));
				}
				//cout << "("<<tem_node->state<<")"<<",";
				cout << ",";
				tem_node = tem_node->next;
			}
			if (flag) {
				cout << "}" << endl;
				flag = false;
			}
		}
	}
	cout << "结果输出结束 count: " << ama->count << endl;
}
