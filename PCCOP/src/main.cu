#include "abpdspre.cuh"
#include "common.cuh"
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;
#define QUEUEBASESIZE 5
#define DEFAULT_XML_FILE "/home/chuancy/git/GPU-CTLPDS/xmlAPDSGenerate/abpds.xml"
#define ARGSNUM 10
#define THREADPERNUM 32
#define BLOCKSIZE 1
AMA *ama_1, *ama_2;
void add_initTrans_to_GQueue_AMA(AMA *ama, Pool *pool) {
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		for (int j = 0; j < abpds_info->stack_size; j++) {
			Trans new_t = { finalStateArray[i], j, -1 };
			//add_one_to_queue(new_t);
			insertTransToAMA(new_t, ama, pool);
		}
	}
}

bool isReach(AMA *ama, Config init_config) {
	int pos1 = init_config.controlLocation * abpds_info->stack_size
			+ init_config.stack1;
	AMANode *node = ama->list[pos1].head.next;
	if (init_config.stack2 == 0 && node != NULL) {
		return true;
	}
	while (node != NULL) {
		if (node->state == -1) {
			return true;
		}
		node->state = node->state & STATEMASK;
		int pos2 = node->state * abpds_info->stack_size + init_config.stack2;
		AMANode *node2 = ama->list[pos2].head.next;
		if (node2 != NULL) {
			return true;
		}
		node = node->next;
	}
	return false;
}

void add_Epsilon_to_queue(AMA *ama) {
	add_to_TMP(ama);
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		for (int j = 0; j < abpds_info->stack_size; j++) {
			int pos = i * abpds_info->stack_size + j;
			AMANode *node = ama->list[pos].head.next;
			while (node != NULL) {
				Trans new_t = { finalStateArray[i], j, node->state };
				add_one_to_queue(new_t);
				node = node->next;
			}
		}
	}
}

int main() {

	char * file_name = DEFAULT_XML_FILE;
	if (parse_abpds_xml(file_name) != 0) {
		printf("Failed to parse abpds\n ");
	} else {
		printf("parse abpds compelet\n");
	}
	//print_parse_result();
	//printStateMap();
	//printStackMap();
	//添加初始格局
	map<string, int>::iterator it_find;
	string init_state, init_stack1, init_stack2;
	int init_state_id, init_stack1_id, init_stack2_id;
	init_state = "p0";
	init_stack1 = "r0";
	init_stack2 = "r3";
	it_find = state_mp.find(init_state);
	if (it_find != state_mp.end()) {
		init_state_id = it_find->second;
	}
	it_find = stack_mp.find(init_stack1);
	if (it_find != stack_mp.end()) {
		init_stack1_id = it_find->second;
	}
	it_find = stack_mp.find(init_stack2);
	if (it_find != stack_mp.end()) {
		init_stack2_id = it_find->second;
	}
	Config init_config = { init_state_id, init_stack1_id, init_stack2_id };

	AMA *ama_1, *ama_2;
	Pool *pool_1, *pool_2;
	CUDA_SAFE_CALL(hipMallocManaged(&ama_1, sizeof(AMA)));
	CUDA_SAFE_CALL(hipMallocManaged(&pool_1, sizeof(Pool));)
	CUDA_SAFE_CALL(hipMallocManaged(&ama_2, sizeof(AMA)));
	CUDA_SAFE_CALL(hipMallocManaged(&pool_2, sizeof(Pool)));
	initGQueue(abpds_info->rule_size);

	initAMA(ama_1, pool_1);
	initAMA(ama_2, pool_2);
	initTMP();

	short int *recursion;
	CUDA_SAFE_CALL(hipMallocManaged(&recursion, sizeof(short int)));

	dim3 dimBlock(THREADPERNUM, 1, 1); //一个块中开threadsPerBlock个线程
	dim3 dimGrid(BLOCKSIZE, 1, 1); //一个gird里开blockSize个块

	int *finish;
	CUDA_SAFE_CALL(hipMallocManaged(&finish, sizeof(int)));

	void **kernelArgs_1 = NULL;
	kernelArgs_1 = (void**) malloc(ARGSNUM * sizeof(*kernelArgs_1));

	kernelArgs_1[0] = malloc(sizeof(finish));
	memcpy(kernelArgs_1[0], &finish, sizeof(finish));

	kernelArgs_1[1] = malloc(sizeof(delta));
	memcpy(kernelArgs_1[1], &delta, sizeof(delta));

	kernelArgs_1[2] = malloc(sizeof(ama_1));
	memcpy(kernelArgs_1[2], &ama_1, sizeof(ama_1));

	kernelArgs_1[3] = malloc(sizeof(pool_1));
	memcpy(kernelArgs_1[3], &pool_1, sizeof(pool_1));

	kernelArgs_1[4] = malloc(sizeof(ama_2));
	memcpy(kernelArgs_1[4], &ama_2, sizeof(ama_2));

	kernelArgs_1[5] = malloc(sizeof(pool_2));
	memcpy(kernelArgs_1[5], &pool_2, sizeof(pool_2));

	kernelArgs_1[6] = malloc(sizeof(recursion));
	memcpy(kernelArgs_1[6], &recursion, sizeof(recursion));

	kernelArgs_1[7] = malloc(sizeof(gqueue));
	memcpy(kernelArgs_1[7], &gqueue, sizeof(gqueue));

	kernelArgs_1[8] = malloc(sizeof(abpds_info));
	memcpy(kernelArgs_1[8], &abpds_info, sizeof(abpds_info));

	kernelArgs_1[9] = malloc(sizeof(tmp_ama));
	memcpy(kernelArgs_1[9], &tmp_ama, sizeof(tmp_ama));

	/*偶数次计算*/

	void **kernelArgs_2 = NULL;
	kernelArgs_2 = (void**) malloc(ARGSNUM * sizeof(*kernelArgs_2));

	kernelArgs_2[0] = malloc(sizeof(finish));
	memcpy(kernelArgs_2[0], &finish, sizeof(finish));

	kernelArgs_2[1] = malloc(sizeof(delta));
	memcpy(kernelArgs_2[1], &delta, sizeof(delta));

	kernelArgs_2[2] = malloc(sizeof(ama_2));
	memcpy(kernelArgs_2[2], &ama_2, sizeof(ama_2));

	kernelArgs_2[3] = malloc(sizeof(pool_2));
	memcpy(kernelArgs_2[3], &pool_2, sizeof(pool_2));

	kernelArgs_2[4] = malloc(sizeof(ama_1));
	memcpy(kernelArgs_2[4], &ama_1, sizeof(ama_1));

	kernelArgs_2[5] = malloc(sizeof(pool_1));
	memcpy(kernelArgs_2[5], &pool_1, sizeof(pool_1));

	kernelArgs_2[6] = malloc(sizeof(recursion));
	memcpy(kernelArgs_2[6], &recursion, sizeof(recursion));

	kernelArgs_2[7] = malloc(sizeof(gqueue));
	memcpy(kernelArgs_2[7], &gqueue, sizeof(gqueue));

	kernelArgs_2[8] = malloc(sizeof(abpds_info));
	memcpy(kernelArgs_2[8], &abpds_info, sizeof(abpds_info));

	kernelArgs_2[9] = malloc(sizeof(tmp_ama));
	memcpy(kernelArgs_2[9], &tmp_ama, sizeof(tmp_ama));

	float elapsedTime = 0.0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	*recursion = 0;
	//向queue中添加初始化数据
	add_initTrans_to_GQueue_AMA(ama_1, pool_1);
	(*recursion)++;
	int epsilion_thread_num = abpds_info->state_size / 32 + 1;
//	int update_block_num=abpds_info->state_size;
	int update_thread_num = abpds_info->stack_size * abpds_info->state_size;
	while (true) {
		if ((*recursion) % 2 == 0) {
			printf("%d:\n", (*recursion));
			compute_epsilon<<<epsilion_thread_num, 32>>>(delta, ama_1, pool_1,
					abpds_info, gqueue, recursion);
			hipDeviceSynchronize();

			add_Epsilon_to_queue(ama_2);
			//printGQueue(gqueue);
			hipLaunchCooperativeKernel((void*) compute_pre_on_pds, dimGrid,
					dimBlock, kernelArgs_2);
			hipDeviceSynchronize();
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);

			hipEventElapsedTime(&elapsedTime, start, stop);

			cout << "calculate Time :" << elapsedTime << "ms" << endl;
			ama_1->count = 0;
			updateAMA<<<1, update_thread_num>>>(ama_1, *recursion, pool_1,
					abpds_info);
			hipDeviceSynchronize();
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);

			hipEventElapsedTime(&elapsedTime, start, stop);

			cout << "update Time :" << elapsedTime << "ms" << endl;
			printAMA(ama_1);
		} else {
			printf("%d:\n", (*recursion));
			compute_epsilon<<<epsilion_thread_num, 32>>>(delta, ama_2, pool_2,
					abpds_info, gqueue, recursion);
			hipDeviceSynchronize();

			add_Epsilon_to_queue(ama_1);
			//printGQueue(gqueue);
			hipLaunchCooperativeKernel((void*) compute_pre_on_pds, dimGrid,
					dimBlock, kernelArgs_1);
			hipDeviceSynchronize();
			ama_2->count = 0;
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);

			hipEventElapsedTime(&elapsedTime, start, stop);

			cout << "calculate Time :" << elapsedTime << "ms" << endl;
			updateAMA<<<1, update_thread_num>>>(ama_2, *recursion, pool_2,
					abpds_info);
			hipDeviceSynchronize();
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);

			hipEventElapsedTime(&elapsedTime, start, stop);

			cout << "update Time :" << elapsedTime << "ms" << endl;
			printAMA(ama_2);
		}

		if ((*recursion) >= 3 && isEqual(ama_1, ama_2)) {
			break;
		}
		deleteTMP();
		if ((*recursion) % 2 == 0) {
			deleteAMA(ama_2, pool_2);
		} else {
			deleteAMA(ama_1, pool_1);
		}
		(*recursion)++;
	}
/*	if (isReach(ama_2, init_config)) {
		printf(
				"The ABPDS has an accepting run from the initial configuration\n");
	} else {
		printf(
				"The ABPDS has not an accepting run from the initial configuration\n");
	}*/

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);

	cout << "Time :" << elapsedTime << "ms" << endl;
	return 0;
}
