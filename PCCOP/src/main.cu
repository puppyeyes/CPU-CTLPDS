#include "abpdspre.cuh"
#include "common.cuh"
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;
#define QUEUEBASESIZE 5
#define DEFAULT_XML_FILE "abpds.xml"
#define ARGSNUM 7
#define THREADPERNUM 32
#define BLOCKSIZE 1
AMA *ama_1, *ama_2;
void add_initTrans_to_GQueue_AMA(AMA *ama, Pool *pool) {
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		for (int j = 1; j < abpds_info->stack_size; j++) {
			Trans new_t = { finalStateArray[i], j, -1 };
			add_one_to_queue(new_t);
			if (!isTransInAMA(new_t, ama,abpds_info)) {
				insertTransToAMA(new_t, ama, pool);
			}
		}
	}
}

int main() {

		char * file_name = DEFAULT_XML_FILE;
	 if (parse_abpds_xml(file_name) != 0) {
	 printf("Failed to parse abpds\n ");
	 } else {
	 printf("parse abpds compelet\n");
	 }

	print_parse_result();

	AMA *ama_1, *ama_2;
	Pool *pool;
	hipMallocManaged(&ama_1, sizeof(AMA));
	hipMallocManaged(&pool, sizeof(Pool));
	initGQueue(QUEUEBASESIZE * abpds_info->stack_size);

	initAMA(ama_1, pool);

	dim3 dimBlock(THREADPERNUM, 1, 1); //一个块中开threadsPerBlock个线程
	dim3 dimGrid(BLOCKSIZE, 1, 1); //一个gird里开blockSize个块

	int *finish;
	CUDA_SAFE_CALL(hipMallocManaged(&finish, sizeof(int)));

	void **kernelArgs = NULL;
	kernelArgs = (void**) malloc(ARGSNUM * sizeof(*kernelArgs));

	kernelArgs[0] = malloc(sizeof(finish));
	memcpy(kernelArgs[0], &finish, sizeof(finish));

	kernelArgs[1] = malloc(sizeof(delta));
	memcpy(kernelArgs[1], &delta, sizeof(delta));

	kernelArgs[2] = malloc(sizeof(ama_1));
	memcpy(kernelArgs[2], &ama_1, sizeof(ama_1));

	kernelArgs[3] = malloc(sizeof(finalStateArray));
	memcpy(kernelArgs[3], &finalStateArray, sizeof(finalStateArray));

	kernelArgs[4] = malloc(sizeof(gqueue));
	memcpy(kernelArgs[4], &gqueue, sizeof(gqueue));

	kernelArgs[5] = malloc(sizeof(abpds_info));
	memcpy(kernelArgs[5], &abpds_info, sizeof(abpds_info));

	kernelArgs[6] = malloc(sizeof(pool));
	memcpy(kernelArgs[6], &pool, sizeof(pool));
	int i = 0;
	//向queue中添加初始化数据
	add_initTrans_to_GQueue_AMA(ama_1, pool);
	i++;
	bool isEqual = false;
	int epsilion_thread_num = abpds_info->state_size / 32 + 1;
	compute_epsilon<<<epsilion_thread_num, 32>>>(delta, ama_1, pool, abpds_info,
			gqueue);
	hipDeviceSynchronize();
	printAMA(ama_1);
/*	while (!(i > 2 && isEqual)) {
		//计算epsilon
		int epsilion_thread_num = abpds_info->state_size / 32 + 1;
		compute_epsilon<<<epsilion_thread_num, 32>>>(delta, ama_1, pool, abpds_info,
				gqueue);
		hipDeviceSynchronize();
		//插入p epsilon -->p
		hipLaunchCooperativeKernel((void*) compute_pre_on_pds, dimGrid,
				dimBlock, kernelArgs);
		hipDeviceSynchronize();

		//更新ama，对比ama
		i++;
	}*/

	return 0;
}
