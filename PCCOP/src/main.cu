#include "utility.cuh"
#include "abpdspre.cuh"
#include "common.cuh"

using namespace std;
using namespace cooperative_groups;
#define QUEUEBASESIZE 5
#define DEFAULT_XML_FILE "abpds.xml"
#define ARGSNUM 7
#define THREADPERNUM 32
#define BLOCKSIZE 1

int main() {

	/*	char * file_name = DEFAULT_XML_FILE;
	 if (parse_abpds_xml(file_name) != 0) {
	 printf("Failed to parse abpds\n ");
	 } else {
	 printf("parse abpds compelet\n");
	 }

	 print_parse_result();*/

	initGQueue(QUEUEBASESIZE * abpds_info->stack_size);

	dim3 dimBlock(THREADPERNUM, 1, 1); //一个块中开threadsPerBlock个线程
	dim3 dimGrid(BLOCKSIZE, 1, 1); //一个gird里开blockSize个块

	bool *isReach = false;
	hipMallocManaged(isReach, sizeof(bool));

	void **kernelArgs = NULL;
	kernelArgs = (void**) malloc(ARGSNUM * sizeof(*kernelArgs));
	kernelArgs[0] = malloc(sizeof(bool));
	memcpy(kernelArgs[0], &isReach, sizeof(bool));

	hipLaunchCooperativeKernel((void*) compute_pre_on_pds, dimGrid, dimBlock,
			kernelArgs, NULL, NULL);
	hipDeviceSynchronize();

	return 0;
}
