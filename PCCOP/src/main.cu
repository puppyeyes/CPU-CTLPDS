#include "utility.cuh"
#include "abpdspre.cuh"
#include "common.cuh"
#include <hip/hip_cooperative_groups.h>

using namespace std;
using namespace cooperative_groups;
#define QUEUEBASESIZE 5
#define DEFAULT_XML_FILE "abpds.xml"
#define ARGSNUM 7
#define THREADPERNUM 32
#define BLOCKSIZE 1

int main() {

	/*	char * file_name = DEFAULT_XML_FILE;
	 if (parse_abpds_xml(file_name) != 0) {
	 printf("Failed to parse abpds\n ");
	 } else {
	 printf("parse abpds compelet\n");
	 }

	 print_parse_result();*/

	AMA ama_1,ama_2;

	initGQueue(QUEUEBASESIZE * abpds_info->stack_size);

	dim3 dimBlock(THREADPERNUM, 1, 1); //一个块中开threadsPerBlock个线程
	dim3 dimGrid(BLOCKSIZE, 1, 1); //一个gird里开blockSize个块

	bool *isReach;
	CUDA_SAFE_CALL(hipMallocManaged(&isReach, sizeof(bool)));

	void **kernelArgs = NULL;
	kernelArgs = (void**) malloc(ARGSNUM * sizeof(*kernelArgs));

	kernelArgs[0] = malloc(sizeof(isReach));
	memcpy(kernelArgs[0], &isReach, sizeof(isReach));

	kernelArgs[1] = malloc(sizeof(delta));
	memcpy(kernelArgs[1], &isReach, sizeof(delta));

	kernelArgs[2] = malloc(sizeof(ama_1));
	memcpy(kernelArgs[2], &ama_1, sizeof(ama_1));

	kernelArgs[3] = malloc(sizeof(ama_2));
	memcpy(kernelArgs[3], &ama_2, sizeof(ama_2));

	kernelArgs[4] = malloc(sizeof(finalStateArray));
	memcpy(kernelArgs[4], &finalStateArray, sizeof(finalStateArray));

	kernelArgs[5] = malloc(sizeof(gqueue));
	memcpy(kernelArgs[5], &gqueue, sizeof(gqueue));

	kernelArgs[6] = malloc(sizeof(abpds_info));
	memcpy(kernelArgs[6], &abpds_info, sizeof(abpds_info));

	hipLaunchCooperativeKernel((void*) compute_pre_on_pds, dimGrid, dimBlock,
			kernelArgs, NULL, NULL);
	hipDeviceSynchronize();

	return 0;
}
