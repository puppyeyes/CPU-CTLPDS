#include "abpdspre.cuh"
#include "common.cuh"
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;
#define QUEUEBASESIZE 5
#define DEFAULT_XML_FILE "abpds.xml"
#define ARGSNUM 7
#define THREADPERNUM 32
#define BLOCKSIZE 1
AMA *ama_1, *ama_2;
void add_initTrans_to_GQueue_AMA(AMA *ama, Pool *pool) {
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		for (int j = 1; j < abpds_info->stack_size; j++) {
			Trans new_t = { finalStateArray[i], j, -1 };
			add_one_to_queue(new_t);
			if (!isTransInAMA(new_t, ama, abpds_info)) {
				insertTransToAMA(new_t, ama, pool);
			}
		}
	}
}

void add_Epsilon_to_queue(AMA *ama) {
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		for (int j = 0; j < abpds_info->state_size; j++) {
			int pos = i * abpds_info->stack_size + j;
			AMANode *node = ama->list[pos].head.next;
			while (node != NULL) {
				Trans new_t = { i, j, node->state };
				add_one_to_queue(new_t);
				node = node->next;
			}
		}
	}
}

int main() {

	char * file_name = DEFAULT_XML_FILE;
	if (parse_abpds_xml(file_name) != 0) {
		printf("Failed to parse abpds\n ");
	} else {
		printf("parse abpds compelet\n");
	}

	print_parse_result();
	printStateMap();
	printStackMap();

	AMA *ama_1, *ama_2;
	Pool *pool_1, *pool_2;
	CUDA_SAFE_CALL(hipMallocManaged(&ama_1, sizeof(AMA)));
	CUDA_SAFE_CALL(hipMallocManaged(&pool_1, sizeof(Pool));)
	CUDA_SAFE_CALL(hipMallocManaged(&ama_2, sizeof(AMA)));
	CUDA_SAFE_CALL(hipMallocManaged(&pool_2, sizeof(Pool)));
	initGQueue(QUEUEBASESIZE * abpds_info->stack_size);

	initAMA(ama_1, pool_1);
	initAMA(ama_2, pool_2);

	short int *recursion;
	CUDA_SAFE_CALL(hipMallocManaged(&recursion, sizeof(short int)));

	dim3 dimBlock(THREADPERNUM, 1, 1); //一个块中开threadsPerBlock个线程
	dim3 dimGrid(BLOCKSIZE, 1, 1); //一个gird里开blockSize个块

	int *finish;
	CUDA_SAFE_CALL(hipMallocManaged(&finish, sizeof(int)));

	void **kernelArgs = NULL;
	kernelArgs = (void**) malloc(ARGSNUM * sizeof(*kernelArgs));

	kernelArgs[0] = malloc(sizeof(finish));
	memcpy(kernelArgs[0], &finish, sizeof(finish));

	kernelArgs[1] = malloc(sizeof(delta));
	memcpy(kernelArgs[1], &delta, sizeof(delta));

	kernelArgs[2] = malloc(sizeof(ama_1));
	memcpy(kernelArgs[2], &ama_1, sizeof(ama_1));

	kernelArgs[3] = malloc(sizeof(recursion));
	memcpy(kernelArgs[3], &recursion, sizeof(recursion));

	kernelArgs[4] = malloc(sizeof(gqueue));
	memcpy(kernelArgs[4], &gqueue, sizeof(gqueue));

	kernelArgs[5] = malloc(sizeof(abpds_info));
	memcpy(kernelArgs[5], &abpds_info, sizeof(abpds_info));

	kernelArgs[6] = malloc(sizeof(pool_1));
	memcpy(kernelArgs[6], &pool_1, sizeof(pool_1));
	int i = 0;
	//向queue中添加初始化数据
	add_initTrans_to_GQueue_AMA(ama_1, pool_1);
	i++;
	bool isEqual = false;
	int epsilion_thread_num = abpds_info->state_size / 32 + 1;
	compute_epsilon<<<epsilion_thread_num, 32>>>(delta, ama_2, pool_2, abpds_info,
			gqueue,recursion);
	hipDeviceSynchronize();
	//printGQueue(gqueue);
	add_Epsilon_to_queue(ama_1);
	hipLaunchCooperativeKernel((void*) compute_pre_on_pds, dimGrid, dimBlock,
			kernelArgs);
	hipDeviceSynchronize();
	printAMA(ama_1);
	/*	while (!(i > 2 && isEqual)) {
	 //计算epsilon
	 int epsilion_thread_num = abpds_info->state_size / 32 + 1;
	 compute_epsilon<<<epsilion_thread_num, 32>>>(delta, ama_1, pool,
	 abpds_info, gqueue);
	 hipDeviceSynchronize();
	 //插入p epsilon -->p
	 add_Epsilon_to_queue(ama_1);
	 //计算pre*
	 hipLaunchCooperativeKernel((void*) compute_pre_on_pds, dimGrid,
	 dimBlock, kernelArgs);
	 hipDeviceSynchronize();
	 //更新ama，对比ama
	 i++;
	 }*/

	return 0;
}
