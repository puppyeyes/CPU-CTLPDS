#include "abpdspre.cuh"
#include "common.cuh"
#include <hip/hip_cooperative_groups.h>

using namespace std;
using namespace cooperative_groups;
#define QUEUEBASESIZE 5
#define DEFAULT_XML_FILE "abpds.xml"
#define ARGSNUM 9
#define THREADPERNUM 32
#define BLOCKSIZE 1

void add_initTrans_to_GQueue_AMA(AMA *ama,Pool *pool){
	for(int i=0;i<abpds_info->finalStateSize;i++){
		for(int j=1;i<abpds_info->stack_size;j++)
		{
			Trans new_t={finalStateArray[i],j,-1};
			add_one_to_queue(new_t);
			if(isTransInAMA(new_t,ama))
			{
				insertTransToAMA(new_t,ama,pool);
			}
		}
	}
}

int main() {

	/*	char * file_name = DEFAULT_XML_FILE;
	 if (parse_abpds_xml(file_name) != 0) {
	 printf("Failed to parse abpds\n ");
	 } else {
	 printf("parse abpds compelet\n");
	 }

	 print_parse_result();*/

	AMA *ama_1, *ama_2;
	Pool *pool;
	initGQueue(QUEUEBASESIZE * abpds_info->stack_size);

	dim3 dimBlock(THREADPERNUM, 1, 1); //一个块中开threadsPerBlock个线程
	dim3 dimGrid(BLOCKSIZE, 1, 1); //一个gird里开blockSize个块

	int *finish;
	CUDA_SAFE_CALL(hipMallocManaged(&finish, sizeof(int)));

	void **kernelArgs = NULL;
	kernelArgs = (void**) malloc(ARGSNUM * sizeof(*kernelArgs));

	kernelArgs[0] = malloc(sizeof(finish));
	memcpy(kernelArgs[0], &finish, sizeof(finish));

	kernelArgs[1] = malloc(sizeof(delta));
	memcpy(kernelArgs[1], &delta, sizeof(delta));

	kernelArgs[2] = malloc(sizeof(ama_1));
	memcpy(kernelArgs[3], &ama_1, sizeof(ama_1));

	kernelArgs[3] = malloc(sizeof(finalStateArray));
	memcpy(kernelArgs[4], &finalStateArray, sizeof(finalStateArray));

	kernelArgs[4] = malloc(sizeof(gqueue));
	memcpy(kernelArgs[5], &gqueue, sizeof(gqueue));

	kernelArgs[5] = malloc(sizeof(abpds_info));
	memcpy(kernelArgs[6], &abpds_info, sizeof(abpds_info));

	kernelArgs[6] = malloc(sizeof(pool));
	memcpy(kernelArgs[7], &pool, sizeof(pool));
	int i = 0;
	//向queue中添加初始化数据
	add_initTrans_to_GQueue_AMA(ama_1,pool);


	i++;
	bool isEqual = false;
	while (!(i > 2 && isEqual)) {

		hipLaunchCooperativeKernel((void*) compute_pre_on_pds, dimGrid,
				dimBlock, kernelArgs);
		hipDeviceSynchronize();

		//更新ama，对比ama
		i++;
	}

	return 0;
}
