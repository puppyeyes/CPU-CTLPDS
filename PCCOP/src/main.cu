#include "abpdspre.cuh"
#include "common.cuh"
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;
#define QUEUEBASESIZE 5
#define DEFAULT_XML_FILE "abpds.xml"
#define ARGSNUM 8
#define THREADPERNUM 32
#define BLOCKSIZE 1
AMA *ama_1, *ama_2;

void add_initTrans_to_GQueue_AMA(AMA *ama, Pool *pool) {
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		for (int j = 0; j < abpds_info->stack_size; j++) {
			Trans new_t = { finalStateArray[i], j, -1 };
			//add_one_to_queue(new_t);
				insertTransToAMA(new_t, ama, pool);
		}
	}
}

void add_Epsilon_to_queue(AMA *ama) {
	add_to_TMP(ama);
	for (int i = 0; i < abpds_info->finalStateSize; i++) {
		for (int j = 0; j < abpds_info->stack_size; j++) {
			int pos = i * abpds_info->stack_size + j;
			AMANode *node = ama->list[pos].head.next;
			while (node != NULL) {
				Trans new_t = { finalStateArray[i], j, node->state };
				add_one_to_queue(new_t);
				node = node->next;
			}
		}
	}
}

int main() {

	char * file_name = DEFAULT_XML_FILE;
	if (parse_abpds_xml(file_name) != 0) {
		printf("Failed to parse abpds\n ");
	} else {
		printf("parse abpds compelet\n");
	}

	print_parse_result();
	printStateMap();
	printStackMap();

	AMA *ama_1, *ama_2;
	Pool *pool_1, *pool_2;
	CUDA_SAFE_CALL(hipMallocManaged(&ama_1, sizeof(AMA)));
	CUDA_SAFE_CALL(hipMallocManaged(&pool_1, sizeof(Pool));)
	CUDA_SAFE_CALL(hipMallocManaged(&ama_2, sizeof(AMA)));
	CUDA_SAFE_CALL(hipMallocManaged(&pool_2, sizeof(Pool)));
	initGQueue(QUEUEBASESIZE * abpds_info->stack_size);

	initAMA(ama_1, pool_1);
	initAMA(ama_2, pool_2);
	initTMP();

	short int *recursion;
	CUDA_SAFE_CALL(hipMallocManaged(&recursion, sizeof(short int)));

	dim3 dimBlock(THREADPERNUM, 1, 1); //一个块中开threadsPerBlock个线程
	dim3 dimGrid(BLOCKSIZE, 1, 1); //一个gird里开blockSize个块

	int *finish;
	CUDA_SAFE_CALL(hipMallocManaged(&finish, sizeof(int)));

	void **kernelArgs = NULL;
	kernelArgs = (void**) malloc(ARGSNUM * sizeof(*kernelArgs));

	kernelArgs[0] = malloc(sizeof(finish));
	memcpy(kernelArgs[0], &finish, sizeof(finish));

	kernelArgs[1] = malloc(sizeof(delta));
	memcpy(kernelArgs[1], &delta, sizeof(delta));

	kernelArgs[2] = malloc(sizeof(ama_2));
	memcpy(kernelArgs[2], &ama_2, sizeof(ama_2));

	kernelArgs[3] = malloc(sizeof(recursion));
	memcpy(kernelArgs[3], &recursion, sizeof(recursion));

	kernelArgs[4] = malloc(sizeof(gqueue));
	memcpy(kernelArgs[4], &gqueue, sizeof(gqueue));

	kernelArgs[5] = malloc(sizeof(abpds_info));
	memcpy(kernelArgs[5], &abpds_info, sizeof(abpds_info));

	kernelArgs[6] = malloc(sizeof(pool_2));
	memcpy(kernelArgs[6], &pool_2, sizeof(pool_2));

	kernelArgs[7] = malloc(sizeof(tmp_ama));
	memcpy(kernelArgs[7], &tmp_ama, sizeof(tmp_ama));
	int i = 0;
	//向queue中添加初始化数据
	add_initTrans_to_GQueue_AMA(ama_1, pool_1);
	i++;
	bool isEqual = false;
	int epsilion_thread_num = abpds_info->state_size / 32 + 1;
	compute_epsilon<<<epsilion_thread_num, 32>>>(delta, ama_2, pool_2, abpds_info,
			gqueue,recursion);
	hipDeviceSynchronize();

	add_Epsilon_to_queue(ama_1);
	//printGQueue(gqueue);
	hipLaunchCooperativeKernel((void*) compute_pre_on_pds, dimGrid, dimBlock,
			kernelArgs);
	hipDeviceSynchronize();
	printAMA(ama_2);
	/*	while (!(i > 2 && isEqual)) {
	 //计算epsilon
	 int epsilion_thread_num = abpds_info->state_size / 32 + 1;
	 compute_epsilon<<<epsilion_thread_num, 32>>>(delta, ama_1, pool,
	 abpds_info, gqueue);
	 hipDeviceSynchronize();
	 //插入p epsilon -->p
	 add_Epsilon_to_queue(ama_1);
	 //计算pre*
	 hipLaunchCooperativeKernel((void*) compute_pre_on_pds, dimGrid,
	 dimBlock, kernelArgs);
	 hipDeviceSynchronize();
	 //更新ama，对比ama
	 i++;
	 }*/

	return 0;
}
