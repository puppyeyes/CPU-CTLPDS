
#include <hip/hip_runtime.h>
/*

#include "ama.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
using namespace std;
static void CheckCudaErrorAux(const char *, unsigned, const char *,
		cudaError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

  Check the return value of the CUDA runtime API call and exit
  the application if the call has failed.

static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, cudaError_t err) {
	if (err == cudaSuccess)
		return;
	std::cerr << statement << " returned " << cudaGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}


int main(void){
	AMA *ama;
	Pool *pool;
	CUDA_CHECK_RETURN(cudaMallocManaged(&ama, sizeof(AMA)));
	CUDA_CHECK_RETURN(cudaMallocManaged (&ama->list, sizeof(AMAList)*AMASIZE));


	//初始化ama.list
	for (int i = 0; i < AMASIZE; i++) {
		ama->list[i].mutex = 0;
//		CUDA_CHECK_RETURN(cudaMallocManaged (&ama->list[i].tail, sizeof(AMANode)));
		//尾指针指向头结点
		ama->list[i].tail=&(ama->list[i].head);
	}
	CUDA_CHECK_RETURN(cudaMallocManaged(&pool, sizeof(Pool)));
	CUDA_CHECK_RETURN(cudaMallocManaged (&pool->item, sizeof(AMANode) * AMAPOOLSIZE));
	pool->size=AMAPOOLSIZE;
	pool->tail=0;


	Trans t={1,2,1};
	insertTransToAMA(t,ama,pool);
	Trans t2={1,2,2};
	insertTransToAMA(t2,ama,pool);
//	cout<<ama->list[2].head.next->state<<endl;
//	cout<<ama->list[2].head.next->next->state<<endl;

	cout<<isTransInAMA(t2,ama,abpds);
	return 0;
}
*/
